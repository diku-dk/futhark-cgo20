#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h> 

#define MIN(a,b)    (((a) < (b)) ? a : b)
#define MAX(a,b)    (((a) < (b)) ? b : a) 


#define GLB_K_MIN   2

#ifndef RACE_FACT
#define RACE_FACT   32  // = H / (Num_Distinct_Pts)
#endif

#ifndef STRIDE
#define STRIDE      32   // = (Max_Ind_Pt - Min_Ind_Pt) / Num_Distinct_Pts
#endif

#define CLelmsz     16 // how many elements fit on a L2 cache line

#define L2Cache     (1024*1024)
#define L2Fract     0.4

#if 1
  #define RACE_EXPNS MAX(1.0, 0.5 * (((float)RACE_FACT)/CLelmsz) * ( (CLelmsz>STRIDE) ? (CLelmsz/STRIDE) : 1 ) )
#else
  #define CTGRACE     1
  #define SHRINK_FACT (0.5*RACE_FACT) //0.625
  #if CTGRACE
    #define RACE_EXPNS  MAX(1.0, SHRINK_FACT)
  #else
    #define RACE_EXPNS  MAX(1.0, SHRINK_FACT / 16)
  #endif
#endif

#define BLOCK       1024
#define GPU_RUNS    10
#define CPU_RUNS    1

#define INP_LEN     50000000
#define Hmax        1000000
 
#ifndef DEBUG_INFO
#define DEBUG_INFO  1
#endif

#ifndef LOCMEMW_PERTHD
#define LOCMEMW_PERTHD 12
#endif



hipDeviceProp_t prop;
unsigned int HWD;
unsigned int SH_MEM_SZ;
unsigned int BLOCK_SZ;

#define NUM_THREADS(n)  min(n, HWD)

#include "histo-kernels.cu.h"
#include "histo-wrap.cu.h"


int autoLocSubHistoDeg(const AtomicPrim prim_kind, const int Q, const int H, const int N) {
    const int elms_per_block = (N + BLOCK - 1) / BLOCK;
    const int el_size = (prim_kind == XCHG)? 2*sizeof(int) : sizeof(int);
    const int m = min((Q*4 / el_size) * BLOCK, elms_per_block) / H;
    // cooperation level can be define independently as
    //     C = min(H/k, B) for some smallish k, or
    // derived from M as
    //     C = ceil(BLOCK/M)
    //const int coop = (BLOCK + m - 1) / m;
    //printf("COOP LEVEL: %d, subhistogram degree: %d\n", coop, m);
    return min(m, BLOCK);
}

int autoGlbSubHistoDeg(
                const AtomicPrim prim_kind, const int H, const int N, const int T, const int L2
) {
    const int el_size = (prim_kind == XCHG)? 2*sizeof(int) : sizeof(int);
    const float frac  = L2Fract * RACE_EXPNS;
    const float k_max = MIN( frac * (L2 / el_size) / T, ((float)N)/T );
    const float coop  = MIN( (float)T, H/k_max );
    return max(1, (int) (T / coop));
}

void autoGlbChunksSubhists(
                const AtomicPrim prim_kind, const int H, const int N, const int T, const int L2,
                int* M, int* num_chunks ) {
    const int el_size = (prim_kind == XCHG)?
                        2*sizeof(int) : sizeof(int);
    
    const float  optim_k_min = GLB_K_MIN;
    const float  coop  = MIN( (float)T, H/optim_k_min );
    const int    Mdeg  = max(1, (int) (T / coop));
    const size_t totsz = Mdeg * H;
    const size_t L2csz = L2Fract * (L2 / el_size) * RACE_EXPNS;
    const int num_chks = (totsz + L2csz - 1) / L2csz;
    const int Hnew     = (H + num_chks - 1) / num_chks;

    *num_chunks = num_chks;
    *M = autoGlbSubHistoDeg(prim_kind, Hnew, N, T, L2);

    printf( "CHUNKING branch: optim_k_min: %f, coop: %f, Mdeg: %d, Hold: %d, Hnew: %d, num_chunks: %d, M: %d\n"
          , optim_k_min, coop, Mdeg, H, Hnew, *num_chunks, *M );
}

void testLocMemAlignmentProblem(const int H, int* h_input, int* h_histo, int* d_input) {
        
        unsigned long tm_seq = goldSeqHisto(INP_LEN, H, h_input, h_histo);
        printf("Histogram Sequential        took: %lu microsecs\n", tm_seq);

        int histos_per_block = 3*BLOCK/min(H, BLOCK);

        unsigned long tm_cas = locMemHwdAddCoop(CAS, INP_LEN, H, histos_per_block, d_input, h_histo);
        printf("Histogram H=%d Local-Mem CAS with subhisto-degree %d took: %lu microsecs\n", H, histos_per_block, tm_cas);

        histos_per_block = 6*BLOCK/min(H, BLOCK);
        tm_cas = locMemHwdAddCoop(CAS, INP_LEN, H, histos_per_block, d_input, h_histo);
        printf("Histogram H=%d Local-Mem CAS with subhisto-degree %d took: %lu microsecs\n", H, histos_per_block, tm_cas);

        histos_per_block = autoLocSubHistoDeg(CAS, LOCMEMW_PERTHD, H, INP_LEN); 
        tm_cas = locMemHwdAddCoop(CAS, INP_LEN, H, histos_per_block, d_input, h_histo);
        printf("Histogram H=%d Local-Mem CAS with subhisto-degree %d took: %lu microsecs\n", H, histos_per_block, tm_cas);
}



void runLocalMemDataset(int* h_input, int* h_histo, int* d_input) {
    const int num_histos = 5;
    const int num_m_degs = 5;
    const int histo_sizes[num_histos] = { 25, 57, 121, 249, 505 }; //{ 64, 128, 256, 512 };
    //const AtomicPrim atomic_kinds[3] = {ADD, CAS, XCHG};
    const int ks[num_m_degs] = { 0, 1, 3, 6, 33 };
    unsigned long runtimes[3][num_histos][num_m_degs];

    for(int i=0; i<num_histos; i++) {
        const int H = histo_sizes[i];
        const int m_opt = autoLocSubHistoDeg(ADD, LOCMEMW_PERTHD, H, INP_LEN);

        const int min_HB = min(H,BLOCK);
        const int subhisto_degs[num_m_degs] = { 1, BLOCK/min_HB, 3*BLOCK/min_HB, 6*BLOCK/min_HB, m_opt }; 
        //{ m_opt, (8*BLOCK) / min_HB, (4*BLOCK) / min_HB, (1*BLOCK) / min_HB, 1};

        goldSeqHisto(INP_LEN, H, h_input, h_histo);

        for(int j=0; j<num_m_degs; j++) {
            const int histos_per_block = subhisto_degs[j];
            runtimes[0][i][j] = locMemHwdAddCoop(ADD, INP_LEN, H, histos_per_block, d_input, h_histo);
            runtimes[1][i][j] = locMemHwdAddCoop(CAS, INP_LEN, H, histos_per_block, d_input, h_histo);
            runtimes[2][i][j] = locMemHwdAddCoop(XCHG, INP_LEN, H, max(histos_per_block/2,1), d_input, h_histo);
        }
    }

    printTextTab<num_histos,num_m_degs>(runtimes, histo_sizes, ks, RACE_FACT);
    //printLaTex  (runtimes, histo_sizes, ks, RACE_FACT);
}


void runGlobalMemDataset(int* h_input, int* h_histo, int* d_input) {
    const int T = NUM_THREADS(INP_LEN);
    const int num_histos = 7;
    const int num_m_degs = 6;
    const int algn = 1;
    const int histo_sizes[num_histos] = { 1*12*1024-algn,  2*12*1024-algn,  4*12*1024-algn
                                        , 8*12*1024-algn, 16*12*1024-algn, 32*12*1024-algn, 64*12*1024-algn };
    const int subhisto_degs[num_m_degs] = { 1, 2, 4, 6, 8, 33 };    
    unsigned long runtimes[3][num_histos][num_m_degs];

    for(int i=0; i<num_histos; i++) {
        const int H = histo_sizes[i];

        goldSeqHisto(INP_LEN, H, h_input, h_histo);

        for(int j=0; j<num_m_degs; j++) {
            int M,    num_chunks;
            int M_lk, num_chunks_lk;

            if(j == num_m_degs-1) {
                autoGlbChunksSubhists(ADD,  H, INP_LEN, T, L2Cache, &M,    &num_chunks);
                autoGlbChunksSubhists(XCHG, H, INP_LEN, T, L2Cache, &M_lk, &num_chunks_lk);
            } else {
                num_chunks = 1; M = subhisto_degs[j];
                num_chunks_lk = 1; M_lk = (M+1)/2;
            }

            if(j==(num_m_degs-1))
                printf("Our M: %d, num_chunks: %d, for H: %d\n", M, num_chunks, H);

            const int B = 256;
            runtimes[0][i][j] = glbMemHwdAddCoop(ADD,  INP_LEN, H, B, M,    num_chunks,    d_input, h_histo);
            runtimes[1][i][j] = glbMemHwdAddCoop(CAS,  INP_LEN, H, B, M,    num_chunks,    d_input, h_histo);
            runtimes[2][i][j] = glbMemHwdAddCoop(XCHG, INP_LEN, H, B, M_lk, num_chunks_lk, d_input, h_histo);
        }
    }

    printf("Running Histo in Global Mem: RACE_FACT: %d, STRIDE: %d, RACE_EXPNS: %f, L2Cache:%d, L2Fract: %f\n",
           RACE_FACT, STRIDE, RACE_EXPNS, L2Cache, L2Fract);

    printTextTab<num_histos,num_m_degs>(runtimes, histo_sizes, subhisto_degs, RACE_FACT);
    //printLaTex  (runtimes, histo_sizes, subhisto_degs, RACE_FACT);
}

/////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////
int main() {
    // set seed for rand()
    srand(2006);

    { // 0. querry the hardware
        int nDevices;
        hipGetDeviceCount(&nDevices);
  
        hipGetDeviceProperties(&prop, 0);
        HWD = prop.maxThreadsPerMultiProcessor * prop.multiProcessorCount;
        BLOCK_SZ = prop.maxThreadsPerBlock;
        SH_MEM_SZ = prop.sharedMemPerBlock;
        if (DEBUG_INFO) {
            printf("Device name: %s\n", prop.name);
            printf("Number of hardware threads: %d\n", HWD);
            printf("Block size: %d\n", BLOCK_SZ);
            printf("Shared memory size: %d\n", SH_MEM_SZ);
            puts("====");
        }
    }

 
    // 1. allocate host memory for input and histogram
    const unsigned int mem_size_input = sizeof(int) * INP_LEN;
    int* h_input = (int*) malloc(mem_size_input);
    const unsigned int mem_size_histo = sizeof(int) * Hmax;
    int* h_histo = (int*) malloc(mem_size_histo);
 
    // 2. initialize host memory
    randomInit(h_input, INP_LEN);
    zeroOut(h_histo, Hmax);
    
    // 3. allocate device memory for input and copy from host
    int* d_input;
    hipMalloc((void**) &d_input, mem_size_input);
    hipMemcpy(d_input, h_input, mem_size_input, hipMemcpyHostToDevice);
 
#if 0
    { // 5. compute a bunch of histograms
        const int H = 128;
        
        unsigned long tm_seq = goldSeqHisto(INP_LEN, H, h_input, h_histo);
        printf("Histogram Sequential        took: %lu microsecs\n", tm_seq);

        int histos_per_block = BLOCK/32;
        //int histos_per_block = autoLocSubHistoDeg(CAS, 12, H, INP_LEN); 
        unsigned long tm_add = locMemHwdAddCoop(ADD, INP_LEN, H, histos_per_block, d_input, h_histo);
        printf("Histogram Local-Mem ADD with subhisto-degree %d took: %lu microsecs\n", histos_per_block, tm_add);

        unsigned long tm_cas = locMemHwdAddCoop(CAS, INP_LEN, H, histos_per_block, d_input, h_histo);
        printf("Histogram Local-Mem CAS with subhisto-degree %d took: %lu microsecs\n", histos_per_block, tm_cas);

        //coop = optimalCoop(XCHG, 12, H);
        unsigned long tm_xch = locMemHwdAddCoop(XCHG, INP_LEN, H, histos_per_block/2, d_input, h_histo);
        printf("Histogram Local-Mem XCG with subhisto-degree %d took: %lu microsecs\n", histos_per_block, tm_xch);
    }
#endif

#if 0
    { // 5. compute a bunch of histograms
        for(int i=0; i<34; i++)
            testLocMemAlignmentProblem(31+i, h_input, h_histo, d_input);
    }
#endif

#if 0
    {
        const int   H = 12288;
        const float k = 0.001;
        const int   B = 256;
        unsigned long tm_seq = goldSeqHisto(INP_LEN, H, h_input, h_histo);
        printf("Before GPU XCG!\n");
        unsigned long tm_xch = glbMemHwdAddCoop(XCHG, INP_LEN, H, k, B, d_input, h_histo);
        printf("Histogram Global-Mem XCG with subhisto-degree %f took: %lu microsecs\n", k, tm_xch);
    }
#endif


#if 0
    runLocalMemDataset(h_input, h_histo, d_input);
#endif

#if 1
    runGlobalMemDataset(h_input, h_histo, d_input);
#endif
    // 7. clean up memory
    free(h_input);
    free(h_histo);
    hipFree(d_input);
}
