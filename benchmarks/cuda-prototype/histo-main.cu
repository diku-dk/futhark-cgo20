#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h> 

#define MIN(a,b)    (((a) < (b)) ? (a) : (b))
#define MAX(a,b)    (((a) < (b)) ? (b) : (a)) 

#define GPU_KIND    1 // 1 -> RTX2080Ti; 2 -> GTX1050Ti

#if (GPU_KIND==1)
    #define MF 5632
    #define RF 0.75
#else // GPU_KIND==2
    #define MF 1024
    #define RF 0.5
#endif

#define GLB_K_MIN   2

#ifndef RACE_FACT
#define RACE_FACT   32 //32  // = H / (Num_Distinct_Pts)
#endif

#ifndef STRIDE
#define STRIDE      16  // = (Max_Ind_Pt - Min_Ind_Pt) / Num_Distinct_Pts
#endif

#define CLelmsz     16 // how many elements fit on a L2 cache line

#define L2Cache     (MF*1024)
#define L2Fract     0.4

#if 1
  #define CTGRACE     0
  #define RACE_EXPNS MAX(1.0, RF * (((float)RACE_FACT)/CLelmsz) * ( (CLelmsz>STRIDE) ? (CLelmsz/STRIDE) : 1 ) )
#else
  #define CTGRACE     1
  #define SHRINK_FACT (0.75*RACE_FACT) //0.625
  #if CTGRACE
    #define RACE_EXPNS  MAX(1.0, SHRINK_FACT)
  #else
    #define RACE_EXPNS  MAX(1.0, SHRINK_FACT / 16)
  #endif
#endif

#define BLOCK       1024
#define GPU_RUNS    50
#define CPU_RUNS    1

#define INP_LEN     50000000
#define Hmax        4000000
 
#ifndef DEBUG_INFO
#define DEBUG_INFO  1
#endif

#ifndef LOCMEMW_PERTHD
#define LOCMEMW_PERTHD 12
#endif



hipDeviceProp_t prop;
unsigned int HWD;
unsigned int SH_MEM_SZ;
unsigned int BLOCK_SZ;

#define NUM_THREADS(n)  min(n, HWD)

#include "histo-kernels.cu.h"
#include "histo-wrap.cu.h"

void autoLocSubHistoDeg(const AtomicPrim prim_kind, const int H, const int N, int* M, int* num_chunks) {
    const int lmem = LOCMEMW_PERTHD * BLOCK * 4;
    const int elms_per_block = (N + BLOCK - 1) / BLOCK;
    const int el_size = (prim_kind == XCHG)? 3*sizeof(int) : sizeof(int);

    float m_prime = MIN( (lmem*1.0 / el_size), (float)elms_per_block ) / H;

    if (prim_kind == ADD) {
        *M = max(1, min( (int)floor(m_prime), BLOCK ) );
    } else {
        float m = max(1.0, m_prime);
        const float RFC = MIN( (float)RACE_FACT, 32.0*pow(RACE_FACT/32.0, 0.33) );
        float f_prime = (BLOCK*RFC) / (m*m*H);
        float f_lower = (prim_kind==CAS) ? ceil(f_prime) : floor(f_prime);
        const int  f  = max(1, (int)f_lower);
        *M = max(1, min( (int)floor((prim_kind==CAS)? m*f : m_prime*f), BLOCK));

        printf("In computeLocM: prim-kind %d, H %d, result f: %f, m: %f, M: %d\n"
              , prim_kind, H, f_prime, m, *M);
    }
    const int len = lmem / (el_size * (*M));
    *num_chunks = (H + len - 1) / len;
}



void autoLocSubHistoDeg0(const AtomicPrim prim_kind, const int H, const int N, int* M, int* num_chunks) {
    const int lmem = LOCMEMW_PERTHD * BLOCK * 4;
    const int elms_per_block = (N + BLOCK - 1) / BLOCK;
    const int el_size_tot = (prim_kind == XCHG)? 3*sizeof(int) : sizeof(int);
    const int el_size = (prim_kind == XCHG)? 2*sizeof(int) : sizeof(int);

    float m     = MIN( (lmem*1.0 / el_size)    , (float)elms_per_block ) / H;
    float m_tot = MIN( (lmem*1.0 / el_size_tot), (float)elms_per_block ) / H;

    if (prim_kind == ADD) {
        *M = max(1, min( (int)floor(m), BLOCK ) );
    } else {
        m = max(1.0, m);
        const float c = BLOCK / m;
        const float RFC = MIN( (float)RACE_FACT, 32.0*pow(RACE_FACT/32.0, 0.33) );
        float tmp1 = c*RFC / (m * H);
        if (m_tot / m > tmp1) {
            *M = min( (int)MAX(floor(m_tot), 1.0), BLOCK );
        } else {
            float tmp = (prim_kind==CAS) ? ceil(tmp1) : floor(tmp1);
            float f = MAX( 1.0, tmp );
            *M = min( (int) floor(m*f), BLOCK);
        }
        printf("In computeLocM: prim-kind %d, H %d, result f: %f, m: %f, M: %d\n"
              , prim_kind, H, tmp1, m, *M);
    }
    const int len = lmem / (el_size_tot * (*M));
    *num_chunks = (H + len - 1) / len;


    // cooperation level can be define independently as
    //     C = min(H/k, B) for some smallish k, or
    // derived from M as
    //     C = ceil(BLOCK/M)
    //const int coop = (BLOCK + m - 1) / m;
    //printf("COOP LEVEL: %d, subhistogram degree: %d\n", coop, m);
    //return min(m, BLOCK);
}

int autoGlbSubHistoDeg(
                const AtomicPrim prim_kind, const int H, const int N, const int T, const int L2
) {
    const int el_size = (prim_kind == XCHG)? 3*sizeof(int) : sizeof(int);
    const float frac  = L2Fract * RACE_EXPNS;
    const float k_max = MIN( frac * (L2 / el_size) / T, ((float)N)/T );
    const float coop_h = (prim_kind == ADD) ? (2.0*H) / k_max : (1.0*H) / k_max; 
    const float coop  = MIN( (float)T, coop_h );
    return max(1, (int) (T / coop));
}

void autoGlbChunksSubhists0(
                const AtomicPrim prim_kind, const int H, const int N, const int T, const int L2,
                int* M, int* num_chunks ) {
    const int el_size = (prim_kind == XCHG)?
                        3*sizeof(int) : sizeof(int);
    
    const float  optim_k_min = GLB_K_MIN;
    const float  coop  = MIN( (float)T, H/optim_k_min );
    const int    Mdeg  = max(1, (int) (T / coop));
    const size_t totsz = Mdeg * H;
    const size_t L2csz = L2Fract * (L2 / el_size) * RACE_EXPNS;
    const int num_chks = (totsz + L2csz - 1) / L2csz;
    const int Hnew     = (H + num_chks - 1) / num_chks;

    *num_chunks = num_chks;
    *M = autoGlbSubHistoDeg(prim_kind, Hnew, N, T, L2);

    printf( "CHUNKING branch: optim_k_min: %f, coop: %f, Mdeg: %d, Hold: %d, Hnew: %d, num_chunks: %d, M: %d\n"
          , optim_k_min, coop, Mdeg, H, Hnew, *num_chunks, *M );
}

void autoGlbChunksSubhists(
                const AtomicPrim prim_kind, const int H, const int N, const int T, const int L2,
                int* M, int* num_chunks ) {
    const int   el_size = (prim_kind == XCHG)?
                          3*sizeof(int) : sizeof(int);
    const float optim_k_min = GLB_K_MIN;
        
    // first part
    float race_exp = max(1.0, (1.0 * RF * RACE_FACT) / (CLelmsz / el_size) );
    float coop_min = MIN( (float)T, H/optim_k_min );
    const int Mdeg  = max(1, (int) (T / coop_min));
    const int H_chk = ( L2Fract * ((1.0*L2Cache) / el_size) * race_exp ) / Mdeg;
    *num_chunks = (H + H_chk - 1) / H_chk;

    // second part
    const float u = (prim_kind == ADD) ? 2.0 : 1.0;
    const float k_max= MIN( L2Fract * ( (1.0*L2Cache) / el_size ) * race_exp, (float)N ) / T;
    const float coop = MIN( T, (u * H_chk) / k_max );
    *M = max( 1, (int)floor(T/coop) );
     
    printf( "CHUNKING branch: optim_k_min: %f, coop: %f, Mdeg: %d, Hold: %d, Hnew: %d, num_chunks: %d, M: %d\n"
          , optim_k_min, coop_min, Mdeg, H, H_chk, *num_chunks, *M );
}


void runLocalMemDataset(int* h_input, uint32_t* h_histo, int* d_input) {
    const int num_histos = 8;
    const int num_m_degs = 6;
    const int histo_sizes[num_histos] = {25, 121, 505, 2041, 6143, 12287, 24575, 49151};
                                        //{/*25, 121, 505, 1024-7,*/ 2048-7, 4089, 6143, 12287, 24575, 49151};
                                        //{ 25, 57, 121, 249, 505, 1024-7, 4096-7, 12288-1, 24575, 4*12*1024-1 };
                                        //{ 64, 128, 256, 512 };
    //const AtomicPrim atomic_kinds[3] = {ADD, CAS, XCHG};
    const int ks[num_m_degs] = { 0, 1, 3, 6, 9, 33 };
    unsigned long runtimes[3][num_histos][num_m_degs];

    for(int i=0; i<num_histos; i++) {
        const int H = histo_sizes[i];
        int m_opt, num_chunks;
        autoLocSubHistoDeg(ADD, H, INP_LEN, &m_opt, &num_chunks);

        // COSMIN is here: this is tricky to adapt since it stores only the
        //                 subhistos and not the num_chunks factor.
        const int min_HB = min(H,BLOCK);
        const int subhisto_degs[num_m_degs] = { 1, BLOCK/min_HB, 3*BLOCK/min_HB, 6*BLOCK/min_HB, 9*BLOCK/min_HB, m_opt };

        { // FOR ADD
            goldSeqHisto<ADD>(INP_LEN, H, h_input, h_histo);

            for(int j=0; j<num_m_degs; j++) {
              if(j == num_m_degs-1) {
                int histos_per_block, num_chunks;
                autoLocSubHistoDeg(ADD,  H, INP_LEN, &histos_per_block, &num_chunks);
                runtimes[0][i][j] = locMemHwdAddCoop(ADD,  INP_LEN, H, histos_per_block, num_chunks, d_input, h_histo);
              } else {
                const int lmem = LOCMEMW_PERTHD*BLOCK, M = subhisto_degs[j];
                int len = lmem / M, num_chunks = (H + len - 1) / len;
                runtimes[0][i][j] = locMemHwdAddCoop(ADD,  INP_LEN, H, M, num_chunks, d_input, h_histo);
              }
            }
        }

        { // FOR CAS
            goldSeqHisto<CAS>(INP_LEN, H, h_input, h_histo);
            for(int j=0; j<num_m_degs; j++) {
              if(j == num_m_degs-1) {
                int histos_per_block, num_chunks;
                autoLocSubHistoDeg(CAS,  H, INP_LEN, &histos_per_block, &num_chunks);
                runtimes[1][i][j] = locMemHwdAddCoop(CAS,  INP_LEN, H, histos_per_block, num_chunks, d_input, h_histo);
              } else {
                const int lmem = LOCMEMW_PERTHD*BLOCK, M = subhisto_degs[j];
                int len = lmem / M, num_chunks = (H + len - 1) / len;
                runtimes[1][i][j] = locMemHwdAddCoop(CAS,  INP_LEN, H, M, num_chunks, d_input, h_histo);
              }
            }
        }

        { // FOR XHCG
            goldSeqHisto<XCHG>(INP_LEN, H, h_input, h_histo);

            for(int j=0; j<num_m_degs; j++) {
              if(j == num_m_degs-1) {
                int histos_per_block, num_chunks;
                autoLocSubHistoDeg(XCHG, H, INP_LEN, &histos_per_block, &num_chunks);
                runtimes[2][i][j] = locMemHwdAddCoop(XCHG, INP_LEN, H, histos_per_block, num_chunks, d_input, h_histo); 
              } else {
                const int lmem = LOCMEMW_PERTHD*BLOCK, M = subhisto_degs[j];
                int len = lmem / (3*M), num_chunks = (H + len - 1) / len;
                runtimes[2][i][j] = locMemHwdAddCoop(XCHG, INP_LEN, H, M, num_chunks, d_input, h_histo);
              }
            }
        }

    }

    //printTextTab<num_histos,num_m_degs>(runtimes, histo_sizes, ks, RACE_FACT);
    printLaTex<num_histos,num_m_degs>  (runtimes, histo_sizes, ks, RACE_FACT);
}


void runGlobalMemDataset(int* h_input, uint32_t* h_histo, int* d_input) {
    const int B = 256;
    const int T = NUM_THREADS(INP_LEN);
    const int num_histos = 7;
    const int num_m_degs = 6;
    const int algn = 1;
    const int histo_sizes[num_histos] = { 1*12*1024-algn,  2*12*1024-algn,  4*12*1024-algn
                                        , 16*12*1024-algn, 32*12*1024-algn
                                        , 64*12*1024-algn, 128*12*1024-algn };
                                        //{ 1*12*1024-algn,  2*12*1024-algn,  4*12*1024-algn
                                        //, 8*12*1024-algn, 16*12*1024-algn, 32*12*1024-algn
                                        //, 64*12*1024-algn, 128*12*1024-algn };
    const int subhisto_degs[num_m_degs] = { 1, 4, 8, 16, 32, 33 };    
    unsigned long runtimes[3][num_histos][num_m_degs];

    for(int i=0; i<num_histos; i++) {
        const int H = histo_sizes[i];

        { // For ADD
            goldSeqHisto<ADD>(INP_LEN, H, h_input, h_histo);

            for(int j=0; j<num_m_degs; j++) {
                int M_add, num_chunks_add;
                if(j == num_m_degs-1) {
                    autoGlbChunksSubhists(ADD,  H, INP_LEN, T, L2Cache, &M_add, &num_chunks_add);
                } else {
                    num_chunks_add = 1; M_add = subhisto_degs[j];
                }
                if(j==(num_m_degs-1))
                    printf("Our M_add: %d, num_chunks_cas: %d, for H: %d\n", M_add, num_chunks_add, H);

                runtimes[0][i][j] = glbMemHwdAddCoop(ADD,  INP_LEN, H, B, M_add, num_chunks_add, d_input, h_histo);
            }
        }

        { // For CAS
            goldSeqHisto<CAS>(INP_LEN, H, h_input, h_histo);

            for(int j=0; j<num_m_degs; j++) {
                int M_cas, num_chunks_cas;
                if(j == num_m_degs-1) {
                    autoGlbChunksSubhists(CAS,  H, INP_LEN, T, L2Cache, &M_cas, &num_chunks_cas);
                } else {
                    num_chunks_cas = 1; M_cas = subhisto_degs[j];
                }
                if(j==(num_m_degs-1))
                    printf("Our M_cas: %d, num_chunks_cas: %d, for H: %d\n", M_cas, num_chunks_cas, H);

                runtimes[1][i][j] = glbMemHwdAddCoop(CAS,  INP_LEN, H, B, M_cas, num_chunks_cas, d_input, h_histo);
            }
        }

        { // For XCHG
            goldSeqHisto<XCHG>(INP_LEN, H, h_input, h_histo);

            for(int j=0; j<num_m_degs; j++) {
                int M_lck, num_chunks_lck;
                if(j == num_m_degs-1) {
                    autoGlbChunksSubhists(XCHG, H, INP_LEN, T, L2Cache, &M_lck, &num_chunks_lck);
                } else {
                    num_chunks_lck = 1; M_lck = (subhisto_degs[j]+2)/3;
                }
                if(j==(num_m_degs-1))
                    printf("Our M_lck: %d, num_chunks_lck: %d, for H: %d\n", M_lck, num_chunks_lck, H);

                runtimes[2][i][j] = glbMemHwdAddCoop(XCHG, INP_LEN, H, B, M_lck, num_chunks_lck, d_input, h_histo);
            }
        }
    }

    printf("Running Histo in Global Mem: RACE_FACT: %d, STRIDE: %d, RACE_EXPNS: %f, L2Cache:%d, L2Fract: %f\n",
           RACE_FACT, STRIDE, RACE_EXPNS, L2Cache, L2Fract);

    //printTextTab<num_histos,num_m_degs>(runtimes, histo_sizes, subhisto_degs, RACE_FACT);
    printLaTex<num_histos,num_m_degs>(runtimes, histo_sizes, subhisto_degs, RACE_FACT);
}

/////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////
int main() {
    // set seed for rand()
    srand(2006);

    { // 0. querry the hardware
        int nDevices;
        hipGetDeviceCount(&nDevices);
  
        hipGetDeviceProperties(&prop, 0);
        HWD = prop.maxThreadsPerMultiProcessor * prop.multiProcessorCount;
        BLOCK_SZ = prop.maxThreadsPerBlock;
        SH_MEM_SZ = prop.sharedMemPerBlock;
        if (DEBUG_INFO) {
            printf("Device name: %s\n", prop.name);
            printf("Number of hardware threads: %d\n", HWD);
            printf("Block size: %d\n", BLOCK_SZ);
            printf("Shared memory size: %d\n", SH_MEM_SZ);
            puts("====");
        }
    }

 
    // 1. allocate host memory for input and histogram
    const unsigned int mem_size_input = sizeof(int) * INP_LEN;
    int* h_input = (int*) malloc(mem_size_input);
    const unsigned int mem_size_histo = sizeof(int) * Hmax;
    uint32_t* h_histo = (uint32_t*) malloc(mem_size_histo);
 
    // 2. initialize host memory
    randomInit(h_input, INP_LEN);
    zeroOut<uint32_t>(h_histo, Hmax);
    
    // 3. allocate device memory for input and copy from host
    int* d_input;
    hipMalloc((void**) &d_input, mem_size_input);
    hipMemcpy(d_input, h_input, mem_size_input, hipMemcpyHostToDevice);

#if 0
    runLocalMemDataset(h_input, h_histo, d_input);
#endif

#if 1
    runGlobalMemDataset(h_input, h_histo, d_input);
#endif
    // 7. clean up memory
    free(h_input);
    free(h_histo);
    hipFree(d_input);
}
