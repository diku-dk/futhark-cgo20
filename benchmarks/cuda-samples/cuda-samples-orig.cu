#include "hip/hip_runtime.h"
// A single program that is essentially the 'histogram' example from
// the CUDA SDK samples, modified to run independently of that
// environment, and with runtime collection.

#include <sys/time.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_cooperative_groups.h>

int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1)
{
  unsigned int resolution=1000000;
  long int diff = (t2->tv_usec + resolution * t2->tv_sec) - (t1->tv_usec + resolution * t1->tv_sec);
  result->tv_sec = diff / resolution;
  result->tv_usec = diff % resolution;
  return (diff<0);
}

// CUDA Runtime
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
// Common definitions
////////////////////////////////////////////////////////////////////////////////
#define HISTOGRAM64_BIN_COUNT 64
#define HISTOGRAM256_BIN_COUNT 256
#define UINT_BITS 32
typedef unsigned int uint;
typedef unsigned char uchar;

////////////////////////////////////////////////////////////////////////////////
// GPU-specific common definitions
////////////////////////////////////////////////////////////////////////////////
#define LOG2_WARP_SIZE 5U
#define WARP_SIZE (1U << LOG2_WARP_SIZE)

//May change on future hardware, so better parametrize the code
#define SHARED_MEMORY_BANKS 16

//Threadblock size: must be a multiple of (4 * SHARED_MEMORY_BANKS)
//because of the bit permutation of threadIdx.x
#define HISTOGRAM64_THREADBLOCK_SIZE (4 * SHARED_MEMORY_BANKS)

//Warps ==subhistograms per threadblock
#define WARP_COUNT 6

//Threadblock size
#define HISTOGRAM256_THREADBLOCK_SIZE (WARP_COUNT * WARP_SIZE)

//Shared memory per threadblock
#define HISTOGRAM256_THREADBLOCK_MEMORY (WARP_COUNT * HISTOGRAM256_BIN_COUNT)

#define UMUL(a, b) ( (a) * (b) )
#define UMAD(a, b, c) ( UMUL((a), (b)) + (c) )

////////////////////////////////////////////////////////////////////////////////
// Reference CPU histogram
////////////////////////////////////////////////////////////////////////////////
extern "C" void histogram64CPU(
    uint *h_Histogram,
    void *h_Data,
    uint byteCount
);

extern "C" void histogram256CPU(
    uint *h_Histogram,
    void *h_Data,
    uint byteCount
);

////////////////////////////////////////////////////////////////////////////////
// GPU histogram
////////////////////////////////////////////////////////////////////////////////
extern "C" void initHistogram64(void);
extern "C" void initHistogram256(void);
extern "C" void closeHistogram64(void);
extern "C" void closeHistogram256(void);

extern "C" void histogram64(
    uint *d_Histogram,
    void *d_Data,
    uint byteCount
);

extern "C" void histogram256(
    uint *d_Histogram,
    void *d_Data,
    uint byteCount
);

const int numRuns = 100;

int checkCudaErrors(hipError_t code) {
  if(code != hipSuccess) {
    printf("GPU Error: %s\n", hipGetErrorString(code));
    exit(33);
    return -1;
  }
  return 0;
}

namespace cg = cooperative_groups;

////////////////////////////////////////////////////////////////////////////////
// GPU-specific common definitions
////////////////////////////////////////////////////////////////////////////////
//Data type used for input data fetches
typedef uint4 data_t;

//May change on future hardware, so better parametrize the code
#define SHARED_MEMORY_BANKS 16

////////////////////////////////////////////////////////////////////////////////
// Main computation pass: compute gridDim.x partial histograms
////////////////////////////////////////////////////////////////////////////////
//Count a byte into shared-memory storage
inline __device__ void addByte(uchar *s_ThreadBase, uint data)
{
    s_ThreadBase[UMUL(data, HISTOGRAM64_THREADBLOCK_SIZE)]++;
}

//Count four bytes of a word
inline __device__ void addWord(uchar *s_ThreadBase, uint data)
{
    //Only higher 6 bits of each byte matter, as this is a 64-bin histogram
    addByte(s_ThreadBase, (data >>  2) & 0x3FU);
    addByte(s_ThreadBase, (data >> 10) & 0x3FU);
    addByte(s_ThreadBase, (data >> 18) & 0x3FU);
    addByte(s_ThreadBase, (data >> 26) & 0x3FU);
}

__global__ void histogram64Kernel(uint *d_PartialHistograms, data_t *d_Data, uint dataCount)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    //Encode thread index in order to avoid bank conflicts in s_Hist[] access:
    //each group of SHARED_MEMORY_BANKS threads accesses consecutive shared memory banks
    //and the same bytes [0..3] within the banks
    //Because of this permutation block size should be a multiple of 4 * SHARED_MEMORY_BANKS
    const uint threadPos =
        ((threadIdx.x & ~(SHARED_MEMORY_BANKS * 4 - 1)) << 0) |
        ((threadIdx.x & (SHARED_MEMORY_BANKS     - 1)) << 2) |
        ((threadIdx.x & (SHARED_MEMORY_BANKS * 3)) >> 4);

    //Per-thread histogram storage
    __shared__ uchar s_Hist[HISTOGRAM64_THREADBLOCK_SIZE * HISTOGRAM64_BIN_COUNT];
    uchar *s_ThreadBase = s_Hist + threadPos;

    //Initialize shared memory (writing 32-bit words)
#pragma unroll

    for (uint i = 0; i < (HISTOGRAM64_BIN_COUNT / 4); i++)
    {
        ((uint *)s_Hist)[threadIdx.x + i * HISTOGRAM64_THREADBLOCK_SIZE] = 0;
    }

    //Read data from global memory and submit to the shared-memory histogram
    //Since histogram counters are byte-sized, every single thread can't do more than 255 submission
    cg::sync(cta);

    for (uint pos = UMAD(blockIdx.x, blockDim.x, threadIdx.x); pos < dataCount; pos += UMUL(blockDim.x, gridDim.x))
    {
        data_t data = d_Data[pos];
        addWord(s_ThreadBase, data.x);
        addWord(s_ThreadBase, data.y);
        addWord(s_ThreadBase, data.z);
        addWord(s_ThreadBase, data.w);
    }

    //Accumulate per-thread histograms into per-block and write to global memory
    cg::sync(cta);

    if (threadIdx.x < HISTOGRAM64_BIN_COUNT)
    {
        uchar *s_HistBase = s_Hist + UMUL(threadIdx.x, HISTOGRAM64_THREADBLOCK_SIZE);

        uint sum = 0;
        uint pos = 4 * (threadIdx.x & (SHARED_MEMORY_BANKS - 1));

#pragma unroll

        for (uint i = 0; i < (HISTOGRAM64_THREADBLOCK_SIZE / 4); i++)
        {
            sum +=
                s_HistBase[pos + 0] +
                s_HistBase[pos + 1] +
                s_HistBase[pos + 2] +
                s_HistBase[pos + 3];
            pos = (pos + 4) & (HISTOGRAM64_THREADBLOCK_SIZE - 1);
        }

        d_PartialHistograms[blockIdx.x * HISTOGRAM64_BIN_COUNT + threadIdx.x] = sum;
    }
}



////////////////////////////////////////////////////////////////////////////////
// Merge histogram64() output
// Run one threadblock per bin; each threadbock adds up the same bin counter
// from every partial histogram. Reads are uncoalesced, but mergeHistogram64
// takes only a fraction of total processing time
////////////////////////////////////////////////////////////////////////////////
#define MERGE_THREADBLOCK_SIZE 256

__global__ void mergeHistogram64Kernel(
    uint *d_Histogram,
    uint *d_PartialHistograms,
    uint histogramCount
)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    __shared__ uint data[MERGE_THREADBLOCK_SIZE];

    uint sum = 0;

    for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)
    {
        sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM64_BIN_COUNT];
    }

    data[threadIdx.x] = sum;

    for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
    {
        cg::sync(cta);

        if (threadIdx.x < stride)
        {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
    }

    if (threadIdx.x == 0)
    {
        d_Histogram[blockIdx.x] = data[0];
    }
}



////////////////////////////////////////////////////////////////////////////////
// CPU interface to GPU histogram calculator
////////////////////////////////////////////////////////////////////////////////
//histogram64kernel() intermediate results buffer
//MAX_PARTIAL_HISTOGRAM64_COUNT == 32768 and HISTOGRAM64_THREADBLOCK_SIZE == 64
//amounts to max. 480MB of input data
static const uint MAX_PARTIAL_HISTOGRAM64_COUNT = 32768;
static uint *d_PartialHistograms;

//Internal memory allocation
extern "C" void initHistogram64(void)
{
    assert(HISTOGRAM64_THREADBLOCK_SIZE % (4 * SHARED_MEMORY_BANKS) == 0);
    checkCudaErrors(hipMalloc((void **)&d_PartialHistograms, MAX_PARTIAL_HISTOGRAM64_COUNT * HISTOGRAM64_BIN_COUNT * sizeof(uint)));
}

//Internal memory deallocation
extern "C" void closeHistogram64(void)
{
    checkCudaErrors(hipFree(d_PartialHistograms));
}

//Round a / b to nearest higher integer value
inline uint iDivUp(uint a, uint b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Snap a to nearest lower multiple of b
inline uint iSnapDown(uint a, uint b)
{
    return a - a % b;
}

extern "C" void histogram64(
    uint *d_Histogram,
    void *d_Data,
    uint byteCount
)
{
    const uint histogramCount = iDivUp(byteCount, HISTOGRAM64_THREADBLOCK_SIZE * iSnapDown(255, sizeof(data_t)));

    assert(byteCount % sizeof(data_t) == 0);
    assert(histogramCount <= MAX_PARTIAL_HISTOGRAM64_COUNT);

    histogram64Kernel<<<histogramCount, HISTOGRAM64_THREADBLOCK_SIZE>>>(
        d_PartialHistograms,
        (data_t *)d_Data,
        byteCount / sizeof(data_t)
    );

    mergeHistogram64Kernel<<<HISTOGRAM64_BIN_COUNT, MERGE_THREADBLOCK_SIZE>>>(
        d_Histogram,
        d_PartialHistograms,
        histogramCount
    );
}

////////////////////////////////////////////////////////////////////////////////
// Shortcut shared memory atomic addition functions
////////////////////////////////////////////////////////////////////////////////

#define TAG_MASK 0xFFFFFFFFU
inline __device__ void addByte(uint *s_WarpHist, uint data, uint threadTag)
{
    atomicAdd(s_WarpHist + data, 1);
}

inline __device__ void addWord(uint *s_WarpHist, uint data, uint tag)
{
    addByte(s_WarpHist, (data >>  0) & 0xFFU, tag);
    addByte(s_WarpHist, (data >>  8) & 0xFFU, tag);
    addByte(s_WarpHist, (data >> 16) & 0xFFU, tag);
    addByte(s_WarpHist, (data >> 24) & 0xFFU, tag);
}

__global__ void histogram256Kernel(uint *d_PartialHistograms, uint *d_Data, uint dataCount)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    //Per-warp subhistogram storage
    __shared__ uint s_Hist[HISTOGRAM256_THREADBLOCK_MEMORY];
    uint *s_WarpHist= s_Hist + (threadIdx.x >> LOG2_WARP_SIZE) * HISTOGRAM256_BIN_COUNT;

    //Clear shared memory storage for current threadblock before processing
#pragma unroll

    for (uint i = 0; i < (HISTOGRAM256_THREADBLOCK_MEMORY / HISTOGRAM256_THREADBLOCK_SIZE); i++)
    {
        s_Hist[threadIdx.x + i * HISTOGRAM256_THREADBLOCK_SIZE] = 0;
    }

    //Cycle through the entire data set, update subhistograms for each warp
    const uint tag = threadIdx.x << (UINT_BITS - LOG2_WARP_SIZE);

    cg::sync(cta);

    for (uint pos = UMAD(blockIdx.x, blockDim.x, threadIdx.x); pos < dataCount; pos += UMUL(blockDim.x, gridDim.x))
    {
        uint data = d_Data[pos];
        addWord(s_WarpHist, data, tag);
    }

    //Merge per-warp histograms into per-block and write to global memory
    cg::sync(cta);

    for (uint bin = threadIdx.x; bin < HISTOGRAM256_BIN_COUNT; bin += HISTOGRAM256_THREADBLOCK_SIZE)
    {
        uint sum = 0;

        for (uint i = 0; i < WARP_COUNT; i++)
        {
            sum += s_Hist[bin + i * HISTOGRAM256_BIN_COUNT] & TAG_MASK;
        }

        d_PartialHistograms[blockIdx.x * HISTOGRAM256_BIN_COUNT + bin] = sum;
    }
}

////////////////////////////////////////////////////////////////////////////////
// Merge histogram256() output
// Run one threadblock per bin; each threadblock adds up the same bin counter
// from every partial histogram. Reads are uncoalesced, but mergeHistogram256
// takes only a fraction of total processing time
////////////////////////////////////////////////////////////////////////////////
#define MERGE_THREADBLOCK_SIZE 256

__global__ void mergeHistogram256Kernel(
    uint *d_Histogram,
    uint *d_PartialHistograms,
    uint histogramCount
)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();

    uint sum = 0;

    for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)
    {
        sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM256_BIN_COUNT];
    }

    __shared__ uint data[MERGE_THREADBLOCK_SIZE];
    data[threadIdx.x] = sum;

    for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
    {
        cg::sync(cta);

        if (threadIdx.x < stride)
        {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
    }

    if (threadIdx.x == 0)
    {
        d_Histogram[blockIdx.x] = data[0];
    }
}

////////////////////////////////////////////////////////////////////////////////
// Host interface to GPU histogram
////////////////////////////////////////////////////////////////////////////////
//histogram256kernel() intermediate results buffer
static const uint PARTIAL_HISTOGRAM256_COUNT = 240;

//Internal memory allocation
extern "C" void initHistogram256(void)
{
    checkCudaErrors(hipMalloc((void **)&d_PartialHistograms, PARTIAL_HISTOGRAM256_COUNT * HISTOGRAM256_BIN_COUNT * sizeof(uint)));
}

//Internal memory deallocation
extern "C" void closeHistogram256(void)
{
    checkCudaErrors(hipFree(d_PartialHistograms));
}

extern "C" void histogram256(
    uint *d_Histogram,
    void *d_Data,
    uint byteCount
)
{
    assert(byteCount % sizeof(uint) == 0);
    histogram256Kernel<<<PARTIAL_HISTOGRAM256_COUNT, HISTOGRAM256_THREADBLOCK_SIZE>>>(
        d_PartialHistograms,
        (uint *)d_Data,
        byteCount / sizeof(uint)
    );

    mergeHistogram256Kernel<<<HISTOGRAM256_BIN_COUNT, MERGE_THREADBLOCK_SIZE>>>(
        d_Histogram,
        d_PartialHistograms,
        PARTIAL_HISTOGRAM256_COUNT
    );
}


extern "C" void histogram64CPU(
    uint *h_Histogram,
    void *h_Data,
    uint byteCount
)
{
    for (uint i = 0; i < HISTOGRAM64_BIN_COUNT; i++)
        h_Histogram[i] = 0;

    assert(sizeof(uint) == 4 && (byteCount % 4) == 0);

    for (uint i = 0; i < (byteCount / 4); i++)
    {
        uint data = ((uint *)h_Data)[i];
        h_Histogram[(data >>  2) & 0x3FU]++;
        h_Histogram[(data >> 10) & 0x3FU]++;
        h_Histogram[(data >> 18) & 0x3FU]++;
        h_Histogram[(data >> 26) & 0x3FU]++;
    }
}



extern "C" void histogram256CPU(
    uint *h_Histogram,
    void *h_Data,
    uint byteCount
)
{
    for (uint i = 0; i < HISTOGRAM256_BIN_COUNT; i++)
        h_Histogram[i] = 0;

    assert(sizeof(uint) == 4 && (byteCount % 4) == 0);

    for (uint i = 0; i < (byteCount / 4); i++)
    {
        uint data = ((uint *)h_Data)[i];
        h_Histogram[(data >>  0) & 0xFFU]++;
        h_Histogram[(data >>  8) & 0xFFU]++;
        h_Histogram[(data >> 16) & 0xFFU]++;
        h_Histogram[(data >> 24) & 0xFFU]++;
    }
}

int main(int argc, char **argv)
{
    uchar *h_Data;
    uint  *h_HistogramCPU, *h_HistogramGPU;
    uchar *d_Data;
    uint  *d_Histogram;
    int PassFailFlag = 1;
    uint byteCount = 64 * 1048576;

    hipDeviceProp_t deviceProp;
    deviceProp.major = 0;
    deviceProp.minor = 0;

    int dev = 0; // First device.

    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));

    printf("CUDA device [%s] has %d Multi-Processors, Compute %d.%d\n",
           deviceProp.name, deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

    h_Data         = (uchar *)malloc(byteCount);
    h_HistogramCPU = (uint *)malloc(HISTOGRAM256_BIN_COUNT * sizeof(uint));
    h_HistogramGPU = (uint *)malloc(HISTOGRAM256_BIN_COUNT * sizeof(uint));

    srand(2009);

    for (uint i = 0; i < byteCount; i++)
    {
        h_Data[i] = rand() % 256;
    }

    checkCudaErrors(hipMalloc((void **)&d_Data, byteCount));
    checkCudaErrors(hipMalloc((void **)&d_Histogram, HISTOGRAM256_BIN_COUNT * sizeof(uint)));
    checkCudaErrors(hipMemcpy(d_Data, h_Data, byteCount, hipMemcpyHostToDevice));

    struct timeval t_start, t_end, t_diff;
    double elapsed;
    {
        initHistogram64();

        printf("Running 64-bin GPU histogram for %u bytes (%u runs)...\n\n", byteCount, numRuns);

        for (int iter = -1; iter < numRuns; iter++)
        {
            //iter == -1 -- warmup iteration
            if (iter == 0)
            {
                hipDeviceSynchronize();
                gettimeofday(&t_start, NULL);
            }

            histogram64(d_Histogram, d_Data, byteCount);
        }

        hipDeviceSynchronize();
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec+t_diff.tv_usec/1e6f);
        double dAvgSecs = (double)elapsed / (double)numRuns;
        printf("histogram64() time (average) : %.5f sec, %.4f MB/sec\n\n", dAvgSecs, ((double)byteCount * 1.0e-6) / dAvgSecs);
        printf("histogram64, Throughput = %.4f MB/s, Time = %.5f s, Size = %u Bytes, NumDevsUsed = %u, Workgroup = %u\n",
               (1.0e-6 * (double)byteCount / dAvgSecs), dAvgSecs, byteCount, 1, HISTOGRAM64_THREADBLOCK_SIZE);

        {
          FILE* f = fopen("histogram64.seconds", "w");
          fprintf(f, "%f\n", dAvgSecs);
          fclose(f);
        }

        checkCudaErrors(hipMemcpy(h_HistogramGPU, d_Histogram, HISTOGRAM64_BIN_COUNT * sizeof(uint), hipMemcpyDeviceToHost));

        histogram64CPU(
            h_HistogramCPU,
            h_Data,
            byteCount
        );

        for (uint i = 0; i < HISTOGRAM64_BIN_COUNT; i++)
            if (h_HistogramGPU[i] != h_HistogramCPU[i])
            {
                PassFailFlag = 0;
            }

        if (PassFailFlag == 0) {
          printf(" ***64-bin histograms do not match!!!***\n\n");
          exit(1);
        }
        closeHistogram64();
    }

    {
        initHistogram256();

        for (int iter = -1; iter < numRuns; iter++)
        {
            //iter == -1 -- warmup iteration
            if (iter == 0)
            {
                checkCudaErrors(hipDeviceSynchronize());
                gettimeofday(&t_start, NULL);
            }

            histogram256(d_Histogram, d_Data, byteCount);
        }

        hipDeviceSynchronize();
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec+t_diff.tv_usec/1e6f);
        double dAvgSecs = elapsed / (double)numRuns;
        printf("histogram256() time (average) : %.5f sec, %.4f MB/sec\n\n", dAvgSecs, ((double)byteCount * 1.0e-6) / dAvgSecs);
        printf("histogram256, Throughput = %.4f MB/s, Time = %.5f s, Size = %u Bytes, NumDevsUsed = %u, Workgroup = %u\n",
               (1.0e-6 * (double)byteCount / dAvgSecs), dAvgSecs, byteCount, 1, HISTOGRAM256_THREADBLOCK_SIZE);

        checkCudaErrors(hipMemcpy(h_HistogramGPU, d_Histogram, HISTOGRAM256_BIN_COUNT * sizeof(uint), hipMemcpyDeviceToHost));

        histogram256CPU(
            h_HistogramCPU,
            h_Data,
            byteCount
        );

        for (uint i = 0; i < HISTOGRAM256_BIN_COUNT; i++)
            if (h_HistogramGPU[i] != h_HistogramCPU[i])
            {
                PassFailFlag = 0;
            }

        if (PassFailFlag == 0) {
          printf(" ***256-bin histograms do not match!!!***\n\n");
          exit(1);
        }

        {
          FILE* f = fopen("histogram256.seconds", "w");
          fprintf(f, "%f\n", dAvgSecs);
          fclose(f);
        }

        closeHistogram256();
    }

    printf("Shutting down...\n");
    checkCudaErrors(hipFree(d_Histogram));
    checkCudaErrors(hipFree(d_Data));
    free(h_HistogramGPU);
    free(h_HistogramCPU);
    free(h_Data);
}
