#include "cub.cuh"  // or equivalently <cub/device/device_histogram.cuh>
#include "helper.cu.h"

template<class Z>
bool validateZ(Z* A, Z* B, uint32_t sizeAB) {
    for(uint32_t i = 0; i < sizeAB; i++)
      if (A[i] != B[i]){
        printf("INVALID RESULT %d (%d, %d)\n", i, A[i], B[i]);
        return false;
      }
    return true;
}

void randomInitNat(uint32_t* data, const uint32_t size, const uint32_t H) {
    for (int i = 0; i < size; ++i) {
        unsigned long int r = rand();
        data[i] = r % H;
    }
}

void histoGold(uint32_t* data, const uint32_t len, const uint32_t H, uint32_t* histo) {
  for(uint32_t i = 0; i < H; i++) {
    histo[i] = 0;
  }
  for(int i = 0; i < len; i++) {
    uint32_t ind = data[i];
    histo[ind]++;
  }
}

int main (int argc, char * argv[]) {
    if (argc != 3 && argc != 4) {
        printf("Usage: %s <N> <H> [timing-file]\n", argv[0]);
        exit(1);
    }
    const uint32_t N = atoi(argv[1]);
    const uint32_t H = atoi(argv[2]);

    //Allocate and Initialize Host data with random values
    uint32_t* h_data  = (uint32_t*)malloc(N*sizeof(uint32_t));
    uint32_t* h_histo = (uint32_t*)malloc(H*sizeof(uint32_t));
    uint32_t* g_histo = (uint32_t*)malloc(H*sizeof(uint32_t));
    randomInitNat(h_data, N, H);

    histoGold(h_data, N, H, g_histo);

    //Allocate and Initialize Device data
    uint32_t* d_data;
    uint32_t* d_histo;
    cudaSucceeded(hipMalloc((void**) &d_data,  N * sizeof(uint32_t)));
    cudaSucceeded(hipMalloc((void**) &d_histo, H * sizeof(uint32_t)));
    cudaSucceeded(hipMemcpy(d_data, h_data, N * sizeof(uint32_t), hipMemcpyHostToDevice));
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    // CUB histogram version

    // Determine temporary device storage requirements
    hipcub::DeviceHistogram::HistogramEven( d_temp_storage, temp_storage_bytes
                                       , d_data, d_histo, H+1, (uint32_t)0
                                       , H, (int32_t)N);
    cudaCheckError();

    // Allocate temporary storage
    cudaSucceeded(hipMalloc(&d_temp_storage, temp_storage_bytes));
    { // one dry run
        hipcub::DeviceHistogram::HistogramEven( d_temp_storage, temp_storage_bytes
                                           , d_data, d_histo, H+1, (uint32_t)0
                                           , H, (int32_t)N );
        hipDeviceSynchronize();
    }
    cudaCheckError();

    double elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL);

    // Compute histogram: excluding inspector time
    for (uint32_t k=0; k<GPU_RUNS; k++) {
        hipcub::DeviceHistogram::HistogramEven( d_temp_storage, temp_storage_bytes
                                           , d_data, d_histo, H+1, (uint32_t)0
                                           , H, (int32_t)N );
    }
    hipDeviceSynchronize();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / ((double)GPU_RUNS);

    cudaCheckError();

    cudaSucceeded(hipMemcpy (h_histo, d_histo, H*sizeof(uint32_t), hipMemcpyDeviceToHost));
    bool success = validateZ<uint32_t>(g_histo, h_histo, H);

    printf("CUB HDW histogram for N=%d, H=%d runs in: %.2f us\n", N, H, elapsed);

    if (argc == 4) {
      writeRuntime(argv[3], elapsed);
    }

    // Cleanup and closing
    hipFree(d_data); hipFree(d_histo); hipFree(d_temp_storage);
    free(h_data); free(g_histo); free(h_histo);

    return success ? 0 : 1;
}
