#include "cub.cuh"  // or equivalently <cub/device/device_histogram.cuh>
#include "helper.cu.h"

template<class Z>
bool validateZ(Z* A, Z* B, uint32_t sizeAB) {
    for(uint32_t i = 0; i < sizeAB; i++)
      if (A[i] != B[i]){
        printf("INVALID RESULT %d (%d, %d)\n", i, A[i], B[i]);
        return false;
      }
    printf("VALID RESULT!\n");
    return true;
}

void randomInitNat(uint32_t* data, const uint32_t size, const uint32_t H) {
    for (int i = 0; i < size; ++i) {
        unsigned long int r = rand();
        data[i] = r % H;
    }
}

void histoGold(uint32_t* data, const uint32_t len, const uint32_t H, uint32_t* histo) {
  for(uint32_t i = 0; i < H; i++) {
    histo[i] = 0;
  }
  for(int i = 0; i < len; i++) {
    uint32_t ind = data[i];
    histo[ind]++;
  } 
}

int main (int argc, char * argv[]) {
    if (argc != 3) {
        printf("Usage: %s <histogram size> <image size>\n", argv[0]);
        exit(1);
    }
    const uint32_t N = atoi(argv[1]);
    const uint32_t H = atoi(argv[2]);
    printf("Computing for image size: %d and histogram size: %d\n", N, H);

    //Allocate and Initialize Host data with random values
    uint32_t* h_data  = (uint32_t*)malloc(N*sizeof(uint32_t));
    uint32_t* h_histo = (uint32_t*)malloc(H*sizeof(uint32_t));
    uint32_t* g_histo = (uint32_t*)malloc(H*sizeof(uint32_t));
    randomInitNat(h_data, N, H);

    { // golden sequential histogram
        double elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL); 

        histoGold(h_data, N, H, g_histo);

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec); 
        printf("Golden (Sequential) Histogram runs in: %.2f microsecs\n", elapsed);
    }

    //Allocate and Initialize Device data
    uint32_t* d_data;
    uint32_t* d_histo;
    cudaSucceeded(hipMalloc((void**) &d_data,  N * sizeof(uint32_t)));
    cudaSucceeded(hipMalloc((void**) &d_histo, H * sizeof(uint32_t)));
    cudaSucceeded(hipMemcpy(d_data, h_data, N * sizeof(uint32_t), hipMemcpyHostToDevice));
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    // CUB histogram version
    //hipMemset(d_histo, 0, H * sizeof(uint32_t));

    // Determine temporary device storage requirements
    hipcub::DeviceHistogram::HistogramEven( d_temp_storage, temp_storage_bytes
                                       , d_data, d_histo, H+1, (uint32_t)0
                                       , H, (int32_t)N);
    cudaCheckError();

    // Allocate temporary storage
    cudaSucceeded(hipMalloc(&d_temp_storage, temp_storage_bytes));
    { // one dry run
        hipcub::DeviceHistogram::HistogramEven( d_temp_storage, temp_storage_bytes
                                           , d_data, d_histo, H+1, (uint32_t)0
                                           , H, (int32_t)N );
        hipDeviceSynchronize();
    }
    cudaCheckError();

    double elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL); 

    // Compute histogram: excluding inspector time
    for(uint32_t k=0; k<GPU_RUNS; k++) {
        hipcub::DeviceHistogram::HistogramEven( d_temp_storage, temp_storage_bytes
                                           , d_data, d_histo, H+1, (uint32_t)0
                                           , H, (int32_t)N );
    }
    hipDeviceSynchronize();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / ((double)GPU_RUNS);

    cudaCheckError();

    cudaSucceeded(hipMemcpy (h_histo, d_histo, H*sizeof(uint32_t), hipMemcpyDeviceToHost));
    printf("CUB Histogram ... ");
    bool success = validateZ<uint32_t>(g_histo, h_histo, H);

    printf("CUB Histogram runs in: %.2f microsecs\n", elapsed);
    double gigaBytesPerSec = 3 * N * sizeof(uint32_t) * 1.0e-3f / elapsed; 
    printf( "CUB Histogram GBytes/sec = %.2f!\n", gigaBytesPerSec); 

    // Cleanup and closing
    hipFree(d_data); hipFree(d_histo); hipFree(d_temp_storage);
    free(h_data); free(g_histo); free(h_histo);

    return success ? 0 : 1;
}
