#include "cub.cuh"   // or equivalently <cub/device/device_histogram.cuh>

#define GPU_RUNS    50
#define real double

#include "helper-keyValf64Add.cu.h"

struct RealAdd
{
    template <typename T>
    __device__ HIPCUB_RUNTIME_FUNCTION __forceinline__
    T operator()(const T &a, const T &b) const {
        return a + b;
    }
};

double sortRedByKeyCUB( uint32_t* data_keys_in,  real* data_vals_in
                      , real* histo, const uint32_t N, const uint32_t H
) {
    uint32_t* data_keys_out;
    real*     data_vals_out;
    uint32_t* unique_keys;
    uint32_t* num_segments;

    { // allocating stuff
        hipMalloc ((void**) &data_keys_out, N * sizeof(uint32_t));
        hipMalloc ((void**) &data_vals_out, N * sizeof(real));
        hipMalloc ((void**) &unique_keys,   H * sizeof(uint32_t));
        hipMalloc ((void**) &num_segments,  sizeof(uint32_t));
    }

    void * tmp_sort_mem = NULL;
    size_t tmp_sort_len = 0;

    { // sort prelude
        hipcub::DeviceRadixSort::SortPairs	( tmp_sort_mem, tmp_sort_len
                                        , data_keys_in, data_keys_out
                                        , data_vals_in, data_vals_out
                                        , (int)N
                                    );
        hipMalloc(&tmp_sort_mem, tmp_sort_len);
    }

    void * tmp_red_mem = NULL;
    size_t tmp_red_len = 0;
    RealAdd redop;
    

    { // reduce-by-key prelude
        hipcub::DeviceReduce::ReduceByKey  ( tmp_red_mem, tmp_red_len
                                        , data_keys_out, unique_keys
                                        , data_vals_out, histo
                                        , num_segments, redop, (int)N
                                        );
        hipMalloc(&tmp_red_mem, tmp_red_len);
    }

    { // one dry run
        hipcub::DeviceRadixSort::SortPairs	( tmp_sort_mem, tmp_sort_len
                                        , data_keys_in, data_keys_out
                                        , data_vals_in, data_vals_out
                                        , (int)N
                                        );
        hipcub::DeviceReduce::ReduceByKey  ( tmp_red_mem, tmp_red_len
                                        , data_keys_out, unique_keys
                                        , data_vals_out, histo
                                        , num_segments, redop, (int)N
                                        );
        hipDeviceSynchronize();
    }

    // timing
    double elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL); 

    for(int k=0; k<GPU_RUNS; k++) {
        hipcub::DeviceRadixSort::SortPairs ( tmp_sort_mem, tmp_sort_len
                                        , data_keys_in, data_keys_out
                                        , data_vals_in, data_vals_out
                                        , (int)N
                                        );
        hipcub::DeviceReduce::ReduceByKey  ( tmp_red_mem, tmp_red_len
                                        , data_keys_out, unique_keys
                                        , data_vals_out, histo
                                        , num_segments, redop, (int)N
                                        );
    }
    hipDeviceSynchronize();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / ((double)GPU_RUNS);

    hipFree(tmp_sort_mem);
    hipFree(tmp_red_mem);
    hipFree(data_keys_out);
    hipFree(data_vals_out);
    hipFree(unique_keys); 
    hipFree(num_segments);

    return elapsed;
}


int main (int argc, char * argv[]) {
    if(argc != 3) {
        printf("Expects two arguments: the image size and the histogram size! argc:%d\n", argc);
        exit(1);
    }
    const uint32_t N = atoi(argv[1]);
    const uint32_t H = atoi(argv[2]);
    printf("Computing for image size: %d and histogram size: %d\n", N, H);

    //Allocate and Initialize Host data with random values
    uint32_t* h_keys  = (uint32_t*)malloc(N*sizeof(uint32_t));
    real*    h_vals  = (real*)   malloc(N*sizeof(real));
    real*    h_histo = (real*)   malloc(H*sizeof(real));
    real*    g_histo = (real*)   malloc(H*sizeof(real));
    randomInit(h_keys, h_vals, N, H);

    { // golden sequential histogram
        double elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL); 

        histoGold(h_keys, h_vals, N, H, g_histo);

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec); 
        printf("Golden (Sequential) Key-RealValue Add Histogram runs in: %.2f microsecs\n", elapsed);
    }

    //Allocate and Initialize Device data
    uint32_t* d_keys;
    real*    d_vals;
    real*    d_histo;
    hipMalloc ((void**) &d_keys,  N * sizeof(uint32_t));
    hipMalloc ((void**) &d_vals,  N * sizeof(real));
    hipMalloc ((void**) &d_histo, H * sizeof(real));
    hipMemcpy(d_keys, h_keys, N * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_vals, h_vals, N * sizeof(real),    hipMemcpyHostToDevice);

    {
        double elapsed = 
            sortRedByKeyCUB ( d_keys,  d_vals, d_histo, N, H );

        hipMemcpy (h_histo, d_histo, H*sizeof(real), hipMemcpyDeviceToHost);
        printf("CUB Key-RealValue Add Histogram ... ");
        validate(g_histo, h_histo, H);

        printf("CUB Key-RealValue Add Histogram runs in: %.2f microsecs\n", elapsed);
        double gigaBytesPerSec = N * (sizeof(uint32_t) + 3*sizeof(real)) * 1.0e-3f / elapsed; 
        printf("CUB Key-RealValue Add Histogram GBytes/sec = %.2f!\n", gigaBytesPerSec); 
    }

    // Cleanup and closing
    hipFree(d_keys); hipFree(d_vals); hipFree(d_histo);
    free(h_keys);  free(h_vals); free(g_histo); free(h_histo);

    return 0;
}
