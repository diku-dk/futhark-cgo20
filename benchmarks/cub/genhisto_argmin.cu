#include "hip/hip_runtime.h"
#include "cub.cuh"   // or equivalently <cub/device/device_histogram.cuh>
#include "helper.cu.h"

#define real  uint64_t

__device__ __host__ inline static
uint64_t pack64(uint32_t ind, uint32_t val) {
   uint64_t res = ind;
   uint64_t tmp = val;
   tmp = tmp << 32;
   res = res | tmp;
   return res;
}

struct indval {
  uint32_t index;
  uint32_t value;
};

__device__ __host__ inline static
indval unpack64(uint64_t t) {
   const uint64_t MASK32bits = 4294967295;
   indval res;
   res.index = (uint32_t) (t & MASK32bits);
   res.value = (uint32_t) (t >> 32);
   return res;
}

__device__ __host__ inline static
uint64_t argmin(uint64_t v1, uint64_t v2) {
    indval arg1 = unpack64(v1);
    indval arg2 = unpack64(v2);
    uint32_t ind, val;
    if (arg1.value < arg2.value) {
        ind = arg2.index; val = arg2.value;
    } else if (arg1.value > arg2.value) {
        ind = arg1.index; val = arg1.value;
    } else { // arg1.value == arg2.value
        ind = min(arg1.index, arg2.index);
        val = arg1.value;
    }
    return pack64(ind, val);
}

void randomInit(uint32_t* keys, uint64_t* vals, const uint32_t N, const uint32_t H) {
    for (int i = 0; i < N; ++i) {
        uint32_t r = rand();
        uint32_t k = r % H;
        uint64_t v = pack64( (uint32_t)r/64, (uint32_t)r );
        keys[i] = k;
        vals[i] = v;
    }
}

void histoGold(uint32_t* keys, uint64_t* vals, const uint32_t N, const uint32_t H, uint64_t* histo) {
  for(uint32_t i = 0; i < H; i++) {
    histo[i] = 0.0;
  }
  for(int i = 0; i < N; i++) {
    uint32_t ind = keys[i];
    uint64_t v   = vals[i];
    histo[ind]   = argmin(histo[ind], v);
  }
}

bool validate(uint64_t* A, uint64_t* B, uint32_t H) {
    for(int i = 0; i < H; i++)
      if ( A[i] != B[i] ) {
        printf("INVALID RESULT %d (%lu,%lu)\n", i, A[i], B[i]);
        return false;
      }
    printf("VALID RESULT!\n");
    return true;
}

struct ArgMin
{
    __device__ HIPCUB_RUNTIME_FUNCTION __forceinline__
    uint64_t operator()(const uint64_t &a, const uint64_t &b) const {
        return argmin(a, b);
    }
};

double sortRedByKeyCUB( uint32_t* data_keys_in,  uint64_t* data_vals_in
                      , uint64_t* histo, const uint32_t N, const uint32_t H
) {
    uint32_t* data_keys_out;
    uint64_t* data_vals_out;
    uint32_t* unique_keys;
    uint32_t* num_segments;

    int beg_bit = 0;
    int end_bit = ceilLog2(H);

    { // allocating stuff
        hipMalloc ((void**) &data_keys_out, N * sizeof(uint32_t));
        hipMalloc ((void**) &data_vals_out, N * sizeof(uint64_t));
        hipMalloc ((void**) &unique_keys,   H * sizeof(uint32_t));
        hipMalloc ((void**) &num_segments,  sizeof(uint32_t));
    }

    void * tmp_sort_mem = NULL;
    size_t tmp_sort_len = 0;

    { // sort prelude
        hipcub::DeviceRadixSort::SortPairs	( tmp_sort_mem, tmp_sort_len
                                        , data_keys_in, data_keys_out
                                        , data_vals_in, data_vals_out
                                        , (int)N,   beg_bit,  end_bit
                                    );
        hipMalloc(&tmp_sort_mem, tmp_sort_len);
    }
    cudaCheckError();

    void * tmp_red_mem = NULL;
    size_t tmp_red_len = 0;
    ArgMin redop;

    { // reduce-by-key prelude
        hipcub::DeviceReduce::ReduceByKey  ( tmp_red_mem, tmp_red_len
                                        , data_keys_out, unique_keys
                                        , data_vals_out, histo
                                        , num_segments, redop, (int)N
                                        );
        hipMalloc(&tmp_red_mem, tmp_red_len);
    }
    cudaCheckError();

    { // one dry run
        hipcub::DeviceRadixSort::SortPairs	( tmp_sort_mem, tmp_sort_len
                                        , data_keys_in, data_keys_out
                                        , data_vals_in, data_vals_out
                                        , (int)N,   beg_bit,  end_bit
                                        );
        hipcub::DeviceReduce::ReduceByKey  ( tmp_red_mem, tmp_red_len
                                        , data_keys_out, unique_keys
                                        , data_vals_out, histo
                                        , num_segments, redop, (int)N
                                        );
        hipDeviceSynchronize();
    }
    cudaCheckError();

    // timing
    double elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL);

    for(int k=0; k<GPU_RUNS; k++) {
        hipcub::DeviceRadixSort::SortPairs ( tmp_sort_mem, tmp_sort_len
                                        , data_keys_in, data_keys_out
                                        , data_vals_in, data_vals_out
                                        , (int)N,   beg_bit,  end_bit
                                        );
        hipcub::DeviceReduce::ReduceByKey  ( tmp_red_mem, tmp_red_len
                                        , data_keys_out, unique_keys
                                        , data_vals_out, histo
                                        , num_segments, redop, (int)N
                                        );
    }
    hipDeviceSynchronize();
    cudaCheckError();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / ((double)GPU_RUNS);

    hipFree(tmp_sort_mem);
    hipFree(tmp_red_mem);
    hipFree(data_keys_out);
    hipFree(data_vals_out);
    hipFree(unique_keys);
    hipFree(num_segments);

    return elapsed;
}


int main (int argc, char * argv[]) {
    if (argc != 3) {
        printf("Usage: %s <histogram size> <image size>\n", argv[0]);
        exit(1);
    }
    const uint32_t N = atoi(argv[1]);
    const uint32_t H = atoi(argv[2]);
    printf("Computing for image size: %d and histogram size: %d\n", N, H);

    //Allocate and Initialize Host data with random values
    uint32_t* h_keys  = (uint32_t*) malloc(N*sizeof(uint32_t));
    uint64_t* h_vals  = (uint64_t*) malloc(N*sizeof(uint64_t));
    uint64_t* h_histo = (uint64_t*) malloc(H*sizeof(uint64_t));
    uint64_t* g_histo = (uint64_t*) malloc(H*sizeof(uint64_t));
    randomInit(h_keys, h_vals, N, H);

    { // golden sequential histogram
        double elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL);

        histoGold(h_keys, h_vals, N, H, g_histo);

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec); 
        printf("Golden (Sequential) Key-Uint64 ArgMin Histogram runs in: %.2f microsecs\n", elapsed);
    }

    //Allocate and Initialize Device data
    uint32_t* d_keys;
    uint64_t* d_vals;
    uint64_t* d_histo;
    hipMalloc ((void**) &d_keys,  N * sizeof(uint32_t));
    hipMalloc ((void**) &d_vals,  N * sizeof(uint64_t));
    hipMalloc ((void**) &d_histo, H * sizeof(uint64_t));
    hipMemcpy(d_keys, h_keys, N * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_vals, h_vals, N * sizeof(uint64_t), hipMemcpyHostToDevice);

    double elapsed = 
      sortRedByKeyCUB ( d_keys,  d_vals, d_histo, N, H );

    hipMemcpy(h_histo, d_histo, H*sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    cudaCheckError();
    printf("CUB Key-Uint64 ArgMin Histogram ... ");
    bool success = validate(g_histo, h_histo, H);

    printf("CUB Key-Uint64 ArgMin Histogram runs in: %.2f microsecs\n", elapsed);
    double gigaBytesPerSec = N * (sizeof(uint32_t) + 3*sizeof(uint64_t)) * 1.0e-3f / elapsed; 
    printf("CUB Key-Uint64 ArgMin Histogram GBytes/sec = %.2f!\n", gigaBytesPerSec); 

    // Cleanup and closing
    hipFree(d_keys); hipFree(d_vals); hipFree(d_histo);
    free(h_keys);  free(h_vals); free(g_histo); free(h_histo);

    return success ? 0 : 1;
}
