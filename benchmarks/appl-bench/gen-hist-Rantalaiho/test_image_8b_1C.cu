#include "hip/hip_runtime.h"
/*
 *
 *
 *  Created on: 27.6.2011
 *      Author: Teemu Rantalaiho (teemu.rantalaiho@helsinki.fi)
 *
 *
 *  Copyright 2011 - 2012 Teemu Rantalaiho
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *
 * Compile with:
 *
 * nvcc -O4 -arch=<your arch> -I../ test_image_8b_1C.cu -o test_image_8b_1C -lnpp -L /usr/local/cuda/lib64/ 
 *  
 *  or
 *
 * nvcc -O4 -arch=<your arch> -I../ test_image_8b_1C.cu -DNONPP -o test_image_8b_1C
 *
 * Also add -DTHURST to include thrust code-path (requires thrust headers to be found of course)
 *
 * Cosmin, compile with:
 * $ nvcc -O4 -arch=compute_35 -Wno-deprecated-declarations -DNONPP test_image_8b_1C.cu -I../ -o test_image_8b_1C
 */

#ifndef TESTMAXIDX
#define TESTMAXIDX   256      // 256 keys / indices
#endif

#define TEST_IS_POW2 1
#define NRUNS   100

#ifdef THRUST
#define ENABLE_THRUST   1   // Enable thrust-based version also (xform-sort_by_key-reduce_by_key)
#else
#define ENABLE_THRUST   0
#endif

//#define NONPP

#ifndef NONPP
#define ENABLE_NPP      1   // NOTE: In order to link, use: -lnpp -L /usr/local/cuda/lib64/ (or similar)
#else
#define ENABLE_NPP      0
#endif

#ifdef OLDARCH
#define FOR_PRE_FERMI   1
#else
#define FOR_PRE_FERMI   0
#endif


#if ENABLE_NPP
#include <npp.h>
#endif

#include <string>
#include <iostream>
#include <assert.h>

#include "cuda_histogram.h"
/*#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>*/


#if ENABLE_THRUST
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/inner_product.h>
#endif

#include <stdio.h>


int gpuAssert(hipError_t code) {
  if(code != hipSuccess) {
    printf("GPU Error: %s\n", hipGetErrorString(code));
    return -1;
  }
  return 0;
}

static bool csv = false;

// Always return 1 -> normal histogram - each sample has same weight
struct test_xform2
{
  __host__ __device__
  void operator() (uint4* input, int i, int* result_index, int* results, int nresults) const {
    uint4 idata = input[i];
#pragma unroll
    for (int resIdx = 0; resIdx < 4; resIdx++)
    {
        unsigned int data = ((unsigned int*)(&idata))[resIdx];
#if 0
        int r = (data >> 16) & 0xFF;
        int g = (data >>  8) & 0xFF;
        int b = (data >>  0) & 0xFF;
        const float scale = (float)(TESTMAXIDX - 1) / (float)(255 + 255 + 255);
        float x = (float)(r + g + b);
        float result = x * scale;
        *result_index++ = (int)result;
#else
        *result_index++ = (data >> 24) & (TESTMAXIDX-1);
        *result_index++ = (data >> 16) & (TESTMAXIDX-1);//0XFF; //0xFF is 255
        *result_index++ = (data >>  8) & (TESTMAXIDX-1);//0XFF;
        *result_index++ = (data >>  0) & (TESTMAXIDX-1);//0XFF;
#endif
        *results++ = 1;
        *results++ = 1;
        *results++ = 1;
        *results++ = 1;
    }
  }
};

typedef struct image_input_s
{
    uint4* image_data;
    int stride; // in uint4 = 16 bytes
    int height;
} image_input;
// Always return 1 -> normal histogram - each sample has same weight
struct xform2dFun
{
  __host__ __device__
  void operator() (image_input input, int* coords, int* result_index, int* results, int nresults) const {
    int x = coords[0];
    int y = coords[1];
    int index = x + input.stride * y;
    uint4 idata = input.image_data[index];
#pragma unroll
    for (int resIdx = 0; resIdx < 4; resIdx++)
    {
        unsigned int data = ((unsigned int*)(&idata))[resIdx];
        *result_index++ = (data >> 24);
        *result_index++ = (data >> 16) & 0XFF;
        *result_index++ = (data >>  8) & 0XFF;
        *result_index++ = (data >>  0) & 0XFF;
        *results++ = 1;
        *results++ = 1;
        *results++ = 1;
        *results++ = 1;
    }
  }
};




// Always return 1 -> normal histogram - each sample has same weight
struct test_xform_old
{
  __host__ __device__
  void operator() (unsigned int* input, int i, int* result_index, int* results, int nresults) const {
    unsigned int data = input[i];
    *result_index++ = (data >> 24);
    *result_index++ = (data >> 16) & 0XFF;
    *result_index++ = (data >>  8) & 0XFF;
    *result_index++ = (data >>  0) & 0XFF;
    *results++ = 1;
    *results++ = 1;
    *results++ = 1;
    *results++ = 1;
  }
};



struct test_sumfun2 {
  __device__ __host__
  int operator() (int res1, int res2) const{
    return res1 + res2;
  }
};


static void printres (int* res, int nres, const char* descr)
{
    if (descr)
        printf("\n%s:\n", descr);
    if (csv)
    {
      printf("vals = [\n");
      for (int i = 0; i < nres; i++)
          printf("%d\n", res[i]);
      printf("]\n");
    }
    else
    {
      printf("vals = [ ");
      for (int i = 0; i < nres; i++)
          printf(" %d, ", res[i]);
      printf("]\n");
    }
}

static void testHistogram(uint4* INPUT, uint4* hostINPUT, int nPixels,  bool print, bool cpurun, bool npp, void* nppSize, void* nppBuffer, void* nppResBuffer, int width, int height, bool use2d)
{
  int nIndex = TESTMAXIDX;
  test_sumfun2 sumFun;
  test_xform2 transformFun;
  xform2dFun xform2d;
  image_input image;
  image.height = height;
  image.stride = width / 4;
  //test_indexfun2 indexFun;
  int zero = 0;
  {
    if(cpurun){
      int* tmpres = (int*)malloc(sizeof(int) * nIndex);
      int* cpures = tmpres;
      memset(tmpres, 0, sizeof(int) * nIndex);

      if (print)
        printf("\nTest reduce_by_key:\n\n");
      
      
      if (use2d)
      {
            image.image_data = hostINPUT;
            for (int y = 0; y < height; y++)
            for (int x = 0; x < width/4; x++)
            {
                int index[16];
                int tmp[16];
                int coord[2] = { x, y };
                xform2d(image, coord, &index[0], &tmp[0], 16);
                for (int tmpi = 0; tmpi < 16; tmpi++)
                    cpures[index[tmpi]] = sumFun(cpures[index[tmpi]], tmp[tmpi]);
            }
      }
      else
      {
            for (int i = 0; i < (nPixels >> 2); i++)
            {
              int index[16];
              int tmp[16];
              transformFun(hostINPUT, i, &index[0], &tmp[0], 16);
              for (int tmpi = 0; tmpi < 16; tmpi++)
                  cpures[index[tmpi]] = sumFun(cpures[index[tmpi]], tmp[tmpi]);
              //printf("i = %d,  out_index = %d,  out_val = (%.3f, %.3f) \n",i, index, tmp.real, tmp.imag);
            }
      }
      if (print)
      {
          printres(tmpres, nIndex, "CPU results:");
      }
      free(tmpres);
    }

    if (!cpurun && !npp)
    {
        if (!use2d){
#if FOR_PRE_FERMI
            callHistogramKernel<histogram_atomic_inc, 4>((unsigned int*)INPUT, transformFunOld, /*indexFun,*/ sumFun, 0, (nPixels), zero, (int*)nppResBuffer, nIndex, true, 0, nppBuffer);
#else
            callHistogramKernel<histogram_atomic_inc, 16>(INPUT, transformFun, /*indexFun,*/ sumFun, 0, (nPixels >> 2), zero, (int*)nppResBuffer, nIndex, true, 0, nppBuffer);
            //printf("LALALAL\n");
#endif
        }
        else
        {
            image.image_data = INPUT;
            callHistogramKernel2Dim<histogram_atomic_inc, 16>(image, xform2d, sumFun, 0, width/4, 0, height, zero, (int*)nppResBuffer, nIndex, true, 0, nppBuffer);
        }
      	//hipDeviceSynchronize();
        //gpuAssert( hipPeekAtLastError() );
      //hipMemcpy(tmpres, nppResBuffer, sizeof(int) * TESTMAXIDX, hipMemcpyDeviceToHost);
        //gpuAssert( hipPeekAtLastError() );
    }
    else if (npp)
    {
#if ENABLE_NPP
        NppiSize oSizeROI = *(NppiSize*)nppSize;
        Npp8u* pDeviceBuffer = (Npp8u*)nppBuffer;
        nppiHistogramEven_8u_C1R(
            (Npp8u*)INPUT, oSizeROI.width, oSizeROI,
            (Npp32s*)nppResBuffer, TESTMAXIDX + 1, 0, TESTMAXIDX,
            pDeviceBuffer);
        //hipMemcpy(tmpres, nppResBuffer, sizeof(int) * TESTMAXIDX, hipMemcpyDeviceToHost);
#endif
    }
#if 0
    if (print && (!cpurun))
    {
      int* tmpres = (int*)malloc(sizeof(int) * nIndex);
      memset(tmpres, 0, sizeof(int) * nIndex);
      hipMemcpy(tmpres, nppResBuffer, sizeof(int) * TESTMAXIDX, hipMemcpyDeviceToHost);
      printres(tmpres, nIndex, "GPU results:");
      free(tmpres);
    }
#endif

  }
}

#if ENABLE_THRUST

// NOTE: Take advantage here of the fact that this is the classical histogram with all values = 1
// And also that we know before hand the number of indices coming out
static void testHistogramParamThrust(unsigned char* INPUT, int index_0, int index_1, bool print)
{
  int nIndex = TESTMAXIDX;
  int N = index_1 - index_0;
  thrust::device_vector<int> vals_out(nIndex);
  thrust::host_vector<int> h_vals_out(nIndex);
  //thrust::device_vector<int> keys(N);
  thrust::device_ptr<unsigned char> keys(INPUT);
  // Sort the data
  thrust::sort(keys, keys + N);
  // And reduce by key - histogram complete
#if 0
  // Note: This codepath is somewhat slow
  test_sumfun2 mysumfun;
  thrust::device_vector<int> keys_out(nIndex);
  thrust::equal_to<int> binary_pred;
  thrust::reduce_by_key(keys, keys + N, thrust::make_constant_iterator(1), keys_out.begin(), vals_out.begin(), binary_pred, mysumfun);
#else
  // This is taken from the thrust histogram example
  thrust::counting_iterator<int> search_begin(0);
  // Find where are the upper bounds of consecutive keys as indices (ie. partition function)
  thrust::upper_bound(keys, keys + N,
                      search_begin, search_begin + nIndex,
                      vals_out.begin());
// compute the histogram by taking differences of the partition function (cumulative histogram)
  thrust::adjacent_difference(vals_out.begin(), vals_out.end(),
                              vals_out.begin());
#endif
  h_vals_out = vals_out;
  if (print)
      printres(&h_vals_out[0], nIndex, "Thrust results");
}
#endif

void printUsage(void)
{
  printf("\n");
  printf("Test order independent reduce-by-key / histogram algorithm with an image histogram\n\n");
  printf("\tOptions:\n\n");
  printf("\t\t--cpu\t\t Run on CPU serially instead of GPU\n");
  printf("\t\t--print\t\t Print results of algorithm (check validity)\n");
  printf("\t\t--thrust\t Run on GPU but using thrust library\n");
  printf("\t\t--csv\t\t When printing add line-feeds to ease openoffice import...\n");
  printf("\t\t--npp\t\t Use NVIDIA Performance Primitives library (NPP) instead.\n");
  printf("\t\t--3ch\t\t Assume 24bits/pixel interleaved RGB data (default).\n");
  printf("\t\t--4ch\t\t Assume 32bits/pixel interleaved ARGB data.\n");

  printf("\t\t--load <name>\t Use 32-bit texture data\n");
  printf("\t\t--2d\t\t Run histogram using 2d-indexing\n");
}





static void fillInput(int* input, const char* filename, int nPixels, bool ch4, bool header)
{
  FILE* file = fopen(filename, "rb");
  //texture->dataRGBA8888 = NULL;
  if (!file)
  {
      char* tmp = (char*)malloc(strlen(filename) + 10);
      if (tmp)
      {
          char* ptr = tmp;
          strcpy(ptr, "../");
          ptr += 3;
          strcpy(ptr, filename);
          file = fopen(tmp, "rb");
      }
  }
  // Read
  if (file)
  {
      unsigned int* data = (unsigned int*)input;
      if (data)
      {
          int i;
          if (header) fseek(file, 16, SEEK_SET);
          for (i = 0; i < nPixels; i++)
          {
              unsigned int raw = 0;
              int bytesPerPixel = ch4 ? 4 : 3;

              int rsize = fread(&raw, bytesPerPixel, 1, file);
              if (rsize != 1)
              {
                  printf(
                      "Warning: Unexpected EOF in texture %s at idx %d\n",
                      filename, i);
                  break;
              }
              if (ch4)
                  data[i] = raw;
              else
                data[i] = (raw & 0x00FFFFFF) | ((i & 0xFFu) << 24);
/*              r = (raw & 0x00FF0000) >> 16;
              g = (raw & 0x0000FF00) >> 8;
              b = (raw & 0x000000FF) >> 0;
              pixel = 0xFF000000 | (b << 16) | (g << 8) | (r << 0);
              data[i] = pixel;*/
          }
      }
      fclose(file);
  }
}



int main (int argc, char** argv)
{
  int i;

  bool cpu = false;
  bool print = false;
  bool thrust = false;
  bool npp = false;
  bool ch4 = false;
  bool use2d = false;

  const char* name = argv[1]; //"feli.raw"; //"logo_small.raw"; //"tex_h.raw"; //"logo_small.raw";//"feli.raw";

  if(print)
    printUsage();

  for (i = 0; i < argc; i++)
  {
    if (argv[i] && strcmp(argv[i], "--cpu") == 0)
      cpu = true;
    else if (argv[i] && strcmp(argv[i], "--csv") == 0)
      csv = true;
    else if (argv[i] && strcmp(argv[i], "--npp") == 0)
      npp = true;
    else if (argv[i] && strcmp(argv[i], "--print") == 0)
      print = true;
    else if (argv[i] && strcmp(argv[i], "--thrust") == 0)
      thrust = true;
    else if (argv[i] && strcmp(argv[i], "--4ch") == 0)
      ch4 = true;
    else if (argv[i] && strcmp(argv[i], "--3ch") == 0)
      ch4 = false;
    else if (argv[i] && strcmp(argv[i], "--2d") == 0)
      use2d = true;
    else if (argv[i] && strcmp(argv[i], "--load") == 0)
    {
      if (argc > i + 1)
        name = argv[i + 1];
    }
  }
  int width = 0;
  int height = 0;
  int nchannels = 0;
  {

    int nPixels = 0;
    {
      // Portable way to check filesize with C-apis (of course safe only up to 2GB):
      FILE* file = fopen(name, "rb");
      int error = -1;
      long filesize = 0;
      int token = 0;
      if (file)
      {
          // Check header first:
          int check = fread(&token, 4, 1, file);
          if (token == -2999999)
          {
              check += fread(&width, 4, 1, file);
              check += fread(&height, 4, 1, file);
              check += fread(&nchannels, 4, 1, file);
              if (check != 4){
                  printf("Error reading image header!\n");
                  error = -2;
              }
          }
          error = fseek(file, 0, SEEK_END);
      }
      if (error == 0)
      {
        filesize = ftell(file);
        if (token == -2999999) filesize -= 16;
        printf("File: %s, filesize = %ld\n", name, filesize);
        fclose(file);
        if (ch4)
            nPixels = (int)((filesize / 16) << 2);
        else
            nPixels = (int)((filesize / 12) << 2);
        if (nPixels <= 0)
        {
          printf("Filesize is too large or small...Sorry...\n");
          return 1;
        }
      }
      else
      {
        printf("Can't access file: %s, errorcode = %d (man fseek)\n", name, error);
        return error;
      }
    }
    printf("Number of pixels in image: %d\n", nPixels);
    // Allocate keys:
    int* INPUT = NULL;

    int* hostINPUT = (int*)malloc(sizeof(int) * nPixels);
    void* nppBuffer = NULL;
    void* nppResBuffer = NULL;
    void* nppSize = NULL;
#if (ENABLE_NPP == 1)
    NppiSize oSizeROI = {0, 0};
    nppSize = &oSizeROI;
#endif
    assert(hostINPUT);
    if (nchannels == 4) ch4 = true;
    else if (nchannels == 3) ch4 = false;
    fillInput(hostINPUT, name, nPixels, ch4, nchannels > 0);
    if (!cpu)
    {
      hipMalloc(&INPUT, sizeof(int) * nPixels);
      assert(INPUT);
      hipMemcpy(INPUT, hostINPUT, sizeof(int) * nPixels, hipMemcpyHostToDevice);
      hipMalloc(&nppResBuffer, sizeof(int) * TESTMAXIDX);
      hipMemset(nppResBuffer, 0, sizeof(int) * TESTMAXIDX);
    }
    // Create events for timing:
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (npp)
    {
      #if (ENABLE_NPP == 0)
            printf("Sorry - you did not compile with npp-support -bailing out...\n");
            return 2;
      #else
            int nDeviceBufferSize;
            int levelCount = TESTMAXIDX + 1;
            // Start guessing from 4096 and div by two
            if (width == 0)
            {
                width = 4096;
                height = nPixels / width;
                while (width > 128)
                {
                    if (width * height == nPixels)
                        break;
                    width >>= 1;
                    height = nPixels / width;
                }
            }
            oSizeROI.width = width*4;
            oSizeROI.height = height;
            nppiHistogramEvenGetBufferSize_8u_C1R(oSizeROI, levelCount ,&nDeviceBufferSize);
            hipMalloc(&nppBuffer, nDeviceBufferSize);
      #endif
    }
    else
    {
        int zero = 0;
        int tmpbufsize = getHistogramBufSize<histogram_atomic_inc>(zero , (int)TESTMAXIDX);
        hipMalloc(&nppBuffer, tmpbufsize);
    }

    hipDeviceSynchronize();

    // Now start timer - we run on stream 0 (default stream):
    hipEventRecord(start, 0);
    bool fst_print = print;
    for (i = 0; i < NRUNS; i++)
    {
      if (thrust)
      {
        #if ENABLE_THRUST
          testHistogramParamThrust((unsigned char*)INPUT, 0, 4*nPixels, fst_print);
        #else
          printf("\nTest was compiled without thrust support! Find 'ENABLE_THRUST' in source-code!\n\n Exiting...\n");
          break;
        #endif
      }
      else
      {
        testHistogram((uint4*)INPUT, (uint4*)hostINPUT, nPixels, fst_print, cpu, npp, nppSize, nppBuffer, nppResBuffer, width, height, use2d);
      }
      fst_print = false;
      // Run only once all stress-tests
    }
    //hipDeviceSynchronize();
    hipDeviceSynchronize();
    //gpuAssert( hipPeekAtLastError() );

    {
        float t_ms;
        hipEventRecord(stop, 0);
        hipDeviceSynchronize();
        hipEventElapsedTime(&t_ms, start, stop);
        double t = t_ms * 0.001f;
        double GKps = (((double)nPixels * (double)NRUNS * 4.0)) / (t*1.e9);
        printf("Average Runtime per loop iteration: %fs, Thoughput (Gkeys/s): %3f GK/s \n", t/NRUNS, GKps);
    }

    if(!cpu) {
      int nIndex = TESTMAXIDX;
      int* tmpres = (int*)malloc(sizeof(int) * nIndex);
      memset(tmpres, 0, sizeof(int) * nIndex);

      hipMemcpy(tmpres, nppResBuffer, sizeof(int) * TESTMAXIDX, hipMemcpyDeviceToHost);
      if (print)
      {
        printf("Width: %d, Height: %d\n", width, height);
        printres(tmpres, nIndex, "GPU results:");
      }
      free(tmpres);
    }

    if (INPUT) hipFree(INPUT);
    if (hostINPUT) free(hostINPUT);
    if (nppBuffer) hipFree(nppBuffer);
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }
  return 0;
}

