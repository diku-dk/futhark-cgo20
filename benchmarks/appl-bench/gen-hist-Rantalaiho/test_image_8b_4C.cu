#include "hip/hip_runtime.h"
/*
 *
 *
 *  Created on: 27.6.2011
 *      Author: Teemu Rantalaiho (teemu.rantalaiho@helsinki.fi)
 *
 *
 *  Copyright 2011 - 2012 Teemu Rantalaiho
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *
 *
 *
 */

 /*
  *     Note: This compiles by default to include NPP-based version, but not
  *     Thrust based version - as of now, Thrust based version is not even
  *     working correctly.
  *
  *  Compile with:
  *
  *  nvcc -O4 -arch=<your_arch> -I../ test_image_8b_4C.cu -o test_image_8b_4C -lnpp -L /usr/local/cuda/lib64/
  *
  *  or without NPP (NVIDIA Performance Primitives):
  * 
  *  nvcc -O4 -arch=<your_arch> -DNONPP -I../ test_image_8b_4C.cu -o test_image_8b_4C
  *
  */

#ifndef TESTMAXIDX
#define TESTMAXIDX   256      // 256 keys / indices
#endif

#define TEST_IS_POW2 1
#define NRUNS   1000

#ifdef THRUST
#define ENABLE_THRUST   1   // Enable thrust-based version also (xform-sort_by_key-reduce_by_key)
#else
#define ENABLE_THRUST   0
#endif

#ifndef NONPP
#define ENABLE_NPP      1   // NOTE: In order to link, use: -lnpp -L /usr/local/cuda/lib64/ (or similar)
#else
#define ENABLE_NPP      0
#endif

#ifdef OLDARCH
#define FOR_PRE_FERMI   1
#else
#define FOR_PRE_FERMI   0
#endif


#if ENABLE_NPP
#include <npp.h>
#endif

#include <string>
#include <iostream>
#include <assert.h>

#include "cuda_histogram.h"
/*#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>*/


#if ENABLE_THRUST
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/inner_product.h>
#endif

#include <stdio.h>

static bool csv = false;

// Always return 1 -> normal histogram - each sample has same weight
template <int channel>
struct test_xformChannel
{
  __host__ __device__
  void operator() (uint4* input, int i, int* result_index, int* results, int nresults) const {
    uint4 idata = input[i];
#pragma unroll
    for (int resIdx = 0; resIdx < 4; resIdx++)
    {
        /*
         * int r = (data >> 16) & 0xFF;
         * int g = (data >>  8) & 0xFF;
         * int b = (data >>  0) & 0xFF;
         */
        // Extract channel
        unsigned int data = ((unsigned int*)(&idata))[resIdx];
        int result = (data >> (8 * channel)) & (TESTMAXIDX-1); //0xFF;
        *result_index++ = result;
        *results++ = 1;
    }
  }
};

struct test_sumfun2 {
  __device__ __host__
  int operator() (int res1, int res2) const{
    return res1 + res2;
  }
};


static void printresImpl (int* res, int nres, const char* descr)
{
    if (descr)
        printf("\n%s:\n", descr);
    if (csv)
    {
      printf("[\n");
      for (int i = 0; i < nres; i++)
          printf("%d\n", res[i]);
      printf("]\n");
    }
    else
    {
      printf("[ ");
      for (int i = 0; i < nres; i++)
          printf(" %d, ", res[i]);
      printf("]\n");
    }
}

static void printres (int* res, int nres, const char* descr)
{
    if (descr)
        printf("\n%s:\n", descr);
    printresImpl(res, nres/4, "Red channel");
    printresImpl(&res[nres/4], nres/4, "Green channel");
    printresImpl(&res[2*nres/4], nres/4, "Blue channel");
    printresImpl(&res[3*nres/4], nres/4, "Alpha channel");
}

static void testHistogram(uint4* INPUT, uint4* hostINPUT, int nPixels,  bool print, bool cpurun, bool npp, void* nppSize, void** nppBuffers, void* nppResBuffer, hipStream_t* streams)
{
  int nIndex = TESTMAXIDX * 4;
  test_sumfun2 sumFun;
  test_xformChannel<0> redChannel;
  test_xformChannel<1> greenChannel;
  test_xformChannel<2> blueChannel;
  test_xformChannel<3> alphaChannel;

  int* tmpres = (int*)malloc(sizeof(int) * nIndex);
  int* cpures = tmpres;
  int* redres = &tmpres[0];
  int* greenres = &tmpres[TESTMAXIDX];
  int* blueres = &tmpres[TESTMAXIDX*2];
  int* alphares = &tmpres[TESTMAXIDX*3];
  int zero = 0;
  {
    {
      if (print)
        printf("\nTest reduce_by_key:\n\n");
      memset(tmpres, 0, sizeof(int) * nIndex);
      if (cpurun)
        for (int i = 0; i < (nPixels >> 2); i++)
        {
          int index[4];
          int tmp[4];
          redChannel(hostINPUT, i, &index[0], &tmp[0], 4);
          for (int tmpi = 0; tmpi < 4; tmpi++)
              redres[index[tmpi]] = sumFun(redres[index[tmpi]], tmp[tmpi]);
          greenChannel(hostINPUT, i, &index[0], &tmp[0], 4);
          for (int tmpi = 0; tmpi < 4; tmpi++)
              greenres[index[tmpi]] = sumFun(greenres[index[tmpi]], tmp[tmpi]);
          blueChannel(hostINPUT, i, &index[0], &tmp[0], 4);
          for (int tmpi = 0; tmpi < 4; tmpi++)
              blueres[index[tmpi]] = sumFun(blueres[index[tmpi]], tmp[tmpi]);
          alphaChannel(hostINPUT, i, &index[0], &tmp[0], 4);
          for (int tmpi = 0; tmpi < 4; tmpi++)
              alphares[index[tmpi]] = sumFun(alphares[index[tmpi]], tmp[tmpi]);

          //printf("i = %d,  out_index = %d,  out_val = (%.3f, %.3f) \n",i, index, tmp.real, tmp.imag);
        }
      if (print && cpurun)
      {
          printres(cpures, nIndex, "CPU results:");
      }
    }

    if (!cpurun && !npp)
    {
      int* tmpbuf = (int*)nppResBuffer;
      int* redgpures = &tmpbuf[0];
      int* greengpures = &tmpbuf[TESTMAXIDX];
      int* bluegpures = &tmpbuf[TESTMAXIDX*2];
      int* alphagpures = &tmpbuf[TESTMAXIDX*3];
      callHistogramKernel<histogram_atomic_inc, 4>(INPUT, redChannel, /*indexFun,*/ sumFun, 0, (nPixels >> 2), zero, redgpures, TESTMAXIDX, true, streams[0], nppBuffers[0]);
      callHistogramKernel<histogram_atomic_inc, 4>(INPUT, greenChannel, /*indexFun,*/ sumFun, 0, (nPixels >> 2), zero, greengpures, TESTMAXIDX, true, streams[1], nppBuffers[1]);
      callHistogramKernel<histogram_atomic_inc, 4>(INPUT, blueChannel, /*indexFun,*/ sumFun, 0, (nPixels >> 2), zero, bluegpures, TESTMAXIDX, true, streams[2], nppBuffers[2]);
      callHistogramKernel<histogram_atomic_inc, 4>(INPUT, alphaChannel, /*indexFun,*/ sumFun, 0, (nPixels >> 2), zero, alphagpures, TESTMAXIDX, true, streams[3], nppBuffers[3]);
      if(print)
        hipMemcpy(tmpres, nppResBuffer, sizeof(int) * nIndex, hipMemcpyDeviceToHost);
    }
    else if (npp)
    {
#if ENABLE_NPP
        NppiSize oSizeROI = *(NppiSize*)nppSize;
        Npp8u* pDeviceBuffer = (Npp8u*)nppBuffers[0];
        Npp32s* histograms[4] = { (Npp32s*)nppResBuffer, ((Npp32s*)nppResBuffer) + TESTMAXIDX, ((Npp32s*)nppResBuffer) + 2*TESTMAXIDX, ((Npp32s*)nppResBuffer) + 3*TESTMAXIDX };
        int level[4] = { TESTMAXIDX + 1, TESTMAXIDX + 1, TESTMAXIDX + 1, TESTMAXIDX + 1 };
        int lowlevel[4] = { 0, 0, 0, 0};
        int uplevel[4] = { TESTMAXIDX, TESTMAXIDX, TESTMAXIDX, TESTMAXIDX };
        nppiHistogramEven_8u_C4R(
            (Npp8u*)INPUT, oSizeROI.width << 2, oSizeROI,
            histograms, level, lowlevel, uplevel,
            pDeviceBuffer);
        hipMemcpy(tmpres, nppResBuffer, sizeof(int) * nIndex, hipMemcpyDeviceToHost);
#endif
    }

    if (print && (!cpurun))
    {
      printres(tmpres, nIndex, "GPU results:");
    }

  }
  free(tmpres);
}

#if ENABLE_THRUST

// NOTE: Take advantage here of the fact that this is the classical histogram with all values = 1
// And also that we know before hand the number of indices coming out
static void testHistogramParamThrust(unsigned char* INPUT, int index_0, int index_1, bool print)
{
  int nIndex = TESTMAXIDX;
  int N = index_1 - index_0;
  thrust::device_vector<int> vals_out(nIndex);
  thrust::host_vector<int> h_vals_out(nIndex);
  //thrust::device_vector<int> keys(N);
  thrust::device_ptr<unsigned char> keys(INPUT);
  // Sort the data
  thrust::sort(keys, keys + N);
  // And reduce by key - histogram complete
#if 0
  // Note: This codepath is somewhat slow
  test_sumfun2 mysumfun;
  thrust::device_vector<int> keys_out(nIndex);
  thrust::equal_to<int> binary_pred;
  thrust::reduce_by_key(keys, keys + N, thrust::make_constant_iterator(1), keys_out.begin(), vals_out.begin(), binary_pred, mysumfun);
#else
  // This is taken from the thrust histogram example
  thrust::counting_iterator<int> search_begin(0);
  // Find where are the upper bounds of consecutive keys as indices (ie. partition function)
  thrust::upper_bound(keys, keys + N,
                      search_begin, search_begin + nIndex,
                      vals_out.begin());
// compute the histogram by taking differences of the partition function (cumulative histogram)
  thrust::adjacent_difference(vals_out.begin(), vals_out.end(),
                              vals_out.begin());
#endif
  h_vals_out = vals_out;
  if (print)
      printres(&h_vals_out[0], nIndex, "Thrust results");
}
#endif

void printUsage(void)
{
  printf("\n");
  printf("Test order independent reduce-by-key / histogram algorithm with an image histogram\n\n");
  printf("\tOptions:\n\n");
  printf("\t\t--cpu\t\t Run on CPU serially instead of GPU\n");
  printf("\t\t--print\t\t Print results of algorithm (check validity)\n");
  printf("\t\t--thrust\t Run on GPU but using thrust library\n");
  printf("\t\t--csv\t\t When printing add line-feeds to ease openoffice import...\n");
  printf("\t\t--npp\t\t Use NVIDIA Performance Primitives library (NPP) instead.\n");
  printf("\t\t--3ch\t\t Assume 24bits/pixel interleaved RGB data (default).\n");
  printf("\t\t--4ch\t\t Assume 32bits/pixel interleaved ARGB data.\n");
  printf("\t\t--streams\t\t Use CUDA-streams in computation - one per channel.\n");
  printf("\t\t--load <name>\t Use 32-bit texture data s\n");
}


#ifdef FUTHARK
static void printFuthark(int* input, int nPixels) {
    printf("\n[ %di32", input[0]);
    for(unsigned int i=1; i<nPixels; i++) {
        printf(", %d", input[i]);
    }
    printf(" ]\n");
}
#endif

static void fillInput(int* input, const char* filename, int nPixels, bool ch4)
{
  FILE* file = fopen(filename, "rb");
  //texture->dataRGBA8888 = NULL;
  if (!file)
  {
      char* tmp = (char*)malloc(strlen(filename) + 10);
      if (tmp)
      {
          char* ptr = tmp;
          strcpy(ptr, "../");
          ptr += 3;
          strcpy(ptr, filename);
          file = fopen(tmp, "rb");
      }
  }
  // Read
  if (file)
  {
      unsigned int* data = (unsigned int*)input;
      if (data)
      {
          int i;
          for (i = 0; i < nPixels; i++)
          {
              unsigned int raw = 0;
              int bytesPerPixel = ch4 ? 4 : 3;
              int rsize = fread(&raw, bytesPerPixel, 1, file);
              if (rsize != 1)
              {
                  printf(
                      "Warning: Unexpected EOF in texture %s at idx %d\n",
                      filename, i);
                  break;
              }
              if (ch4)
                  data[i] = raw;
              else
                data[i] = (raw & 0x00FFFFFF) | ((i & 0xFFu) << 24);
/*              r = (raw & 0x00FF0000) >> 16;
              g = (raw & 0x0000FF00) >> 8;
              b = (raw & 0x000000FF) >> 0;
              pixel = 0xFF000000 | (b << 16) | (g << 8) | (r << 0);
              data[i] = pixel;*/
          }
      }
      fclose(file);
  }
}


//#define FUTHARK

int main (int argc, char** argv)
{
  int i;

  bool cpu = false;
  bool print = false;
  bool thrust = false;
  bool npp = false;
  bool ch4 = false;
  bool use_streams = false;

  const char* name = argv[1]; //"logo_small.raw"; //"tex_h.raw"; //"logo_small.raw"; //"feli.raw"; //"texture.raw";

#ifndef FUTHARK
  if(print)
    printUsage();
#endif

  for (i = 0; i < argc; i++)
  {
    if (argv[i] && strcmp(argv[i], "--cpu") == 0)
      cpu = true;
    else if (argv[i] && strcmp(argv[i], "--csv") == 0)
      csv = true;
    else if (argv[i] && strcmp(argv[i], "--npp") == 0)
      npp = true;
    else if (argv[i] && strcmp(argv[i], "--print") == 0)
      print = true;
    else if (argv[i] && strcmp(argv[i], "--thrust") == 0)
      thrust = true;
    else if (argv[i] && strcmp(argv[i], "--4ch") == 0)
      ch4 = true;
    else if (argv[i] && strcmp(argv[i], "--3ch") == 0)
      ch4 = false;
    else if (argv[i] && ((strcmp(argv[i], "--streams") == 0) ||
                         (strcmp(argv[i], "--stream")) == 0) )
      use_streams = true;
    else if (argv[i] && strcmp(argv[i], "--load") == 0)
    {
      if (argc > i + 1)
        name = argv[i + 1];
    }
  }

  {

    int nPixels = 0;
    {
      // Portable way to check filesize with C-apis (of course safe only up to 2GB):
      FILE* file = fopen(name, "rb");
      int error = -1;
      long filesize = 0;

      if (file)
        error = fseek(file, 0, SEEK_END);

      if (error == 0)
      {
        filesize = ftell(file);
#ifndef FUTHARK
        printf("File: %s, filesize = %ld\n", name, filesize);
#endif
        fclose(file);
        if (ch4)
            nPixels = (int)((filesize / 16) << 2);
        else
            nPixels = (int)((filesize / 12) << 2);
        if (nPixels <= 0)
        {
          printf("Filesize is too large or small...Sorry...\n");
          return 1;
        }
      }
      else
      {
        printf("Can't access file: %s, errorcode = %d (man fseek)\n", name, error);
        return error;
      }
    }

    printf("Number of pixels in image: %d\n", nPixels);
    // Allocate keys:
    int* INPUT = NULL;
    int* hostINPUT = (int*)malloc(sizeof(int) * nPixels);
    void* nppBuffers[4] = { NULL, NULL, NULL, NULL };
    void* nppResBuffer = NULL;
    void* nppSize = NULL;
    hipStream_t streams[4] = { 0, 0, 0, 0 };
#if (ENABLE_NPP == 1)
    NppiSize oSizeROI = {0, 0};
    nppSize = &oSizeROI;
#endif
    assert(hostINPUT);
    fillInput(hostINPUT, name, nPixels, ch4);

#ifdef FUTHARK
    // print to Futhark
    printFuthark(hostINPUT, nPixels);
    exit(0);
#endif

    //printf("Cosmin: Num Pixels: %d\n", nPixels);

    if (!cpu)
    {
      hipMalloc(&INPUT, sizeof(int) * nPixels);
      assert(INPUT);
      hipMemcpy(INPUT, hostINPUT, sizeof(int) * nPixels, hipMemcpyHostToDevice);
      hipMalloc(&nppResBuffer, sizeof(int) * TESTMAXIDX * 4);
      hipMemset(nppResBuffer, 0, sizeof(int) * TESTMAXIDX * 4);
    }
    // Create events for timing:
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (use_streams)
    {
        for (int sid = 0; sid < 4; sid++)
          hipStreamCreate(&streams[sid]);
    }


    if (npp)
    {
      #if (ENABLE_NPP == 0)
            printf("Sorry - you did not compile with npp-support -bailing out...\n");
            return 2;
      #else
            int nDeviceBufferSize;
            int levelCounts[] = {TESTMAXIDX + 1, TESTMAXIDX + 1, TESTMAXIDX + 1, TESTMAXIDX + 1 };
            // Start guessing from 4096 and div by two
            int width = 4096;
            int height = nPixels / width;
            while (width > 128)
            {
                if (width * height == nPixels)
                    break;
                width >>= 1;
                height = nPixels / width;
            }
            oSizeROI.width = width;
            oSizeROI.height = height;
            nppiHistogramEvenGetBufferSize_8u_C4R(oSizeROI, levelCounts ,&nDeviceBufferSize);
            hipMalloc(&nppBuffers[0], nDeviceBufferSize);
      #endif
    }
    else
    {
        int zero = 0;
        int tmpbufsize = getHistogramBufSize<histogram_atomic_inc>(zero , (int)(TESTMAXIDX));
        for (int sid = 0; sid < 4; sid++ )
          hipMalloc(&nppBuffers[sid], tmpbufsize);
    }

    // Now start timer - we run on stream 0 (default stream):
    hipEventRecord(start, 0);

    for (i = 0; i < NRUNS; i++)
    {
      if (thrust)
      {
        #if ENABLE_THRUST
          testHistogramParamThrust((unsigned char*)INPUT, 0, 4*nPixels, print);
        #else
          printf("\nTest was compiled without thrust support! Find 'ENABLE_THRUST' in source-code!\n\n Exiting...\n");
          break;
        #endif
      }
      else
      {
        testHistogram((uint4*)INPUT, (uint4*)hostINPUT, nPixels, print, cpu, npp, nppSize, nppBuffers, nppResBuffer, &streams[0]);
      }
      print = false;
      // Run only once all stress-tests
    }
    
    {
        float t_ms;
        hipEventRecord(stop, 0);
        hipDeviceSynchronize();
        hipEventElapsedTime(&t_ms, start, stop);
        double t = t_ms * 0.001f;
        double GKps = (((double)nPixels * (double)NRUNS * 4.0)) / (t*1.e9);
        printf("Average Runtime per loop iteration: %fs, Thoughput (Gkeys/s): %3f GK/s \n", t/NRUNS, GKps);
    }

    if (INPUT) hipFree(INPUT);
    if (hostINPUT) free(hostINPUT);
    for (int sid = 0; sid < 4; sid++) if (nppBuffers[sid]) hipFree(nppBuffers[sid]);
    if (use_streams) for (int sid = 0; sid < 4; sid++) hipStreamDestroy(streams[sid]);
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }
  return 0;
}

